#include "THCApply.cuh"

static inline int curGPU() {
  int curDev;
  THCudaCheck(hipGetDevice(&curDev));
  return curDev;
}

THC_API void
THCudaTensor_copy(THCState* state, THCudaTensor* dst, THCudaTensor* src) {
  long totalElements = THCudaTensor_nElement(state, dst);

  THArgCheck(totalElements == THCudaTensor_nElement(state, src), 2,
             "sizes do not match");

  if (THCudaTensor_nDimension(state, dst) == 0) {
    // Zero-dim tensor; copy nothing
    return;
  }

  // We can memcpy the memory if:
  // -both tensors are contiguous; or,
  // -there is only one element to copy; or,
  // -FIXME: if both tensors have matching size and stride arrays, and no
  // holes within (in other words, there is some permutation that can be applied
  // to the size/strides such that the resulting tensor is contiguous).
  bool srcContig = THCudaTensor_isContiguous(state, src);
  bool dstContig = THCudaTensor_isContiguous(state, dst);
  bool memcpyEligible = (srcContig && dstContig) || (totalElements == 1);

  int oldDev = curGPU();
  int srcDev = THCudaTensor_getDevice(state, src);
  int dstDev = THCudaTensor_getDevice(state, dst);

  // empirically, running the kernel on the device that holds the
  // non-contiguous tensor is faster by 5-10x
  int copyDev   = dstContig ? srcDev : dstDev;
  int remoteDev = dstContig ? dstDev : srcDev;

  if (srcDev == dstDev) {
    if (oldDev != srcDev) {
      THCudaCheck(hipSetDevice(srcDev));
    }
  } else {
    // synchronize remote device before copy
    hipEvent_t dataReady;
    THCudaCheck(hipSetDevice(remoteDev));
    THCudaCheck(hipEventCreate(&dataReady));
    THCudaCheck(hipEventRecord(
                  dataReady,
                  THCState_getDeviceStream(state, remoteDev, THCState_getCurrentStreamIndex(state))));
    THCudaCheck(hipSetDevice(copyDev));
    THCudaCheck(hipStreamWaitEvent(
                  THCState_getDeviceStream(state, copyDev, THCState_getCurrentStreamIndex(state)),
                  dataReady, 0));
    THCudaCheck(hipEventDestroy(dataReady));
  }

  if (memcpyEligible) {
    THCudaCheck(hipMemcpyAsync(THCudaTensor_data(state, dst),
                                THCudaTensor_data(state, src),
                                totalElements * sizeof(float),
                                hipMemcpyDeviceToDevice,
                                THCState_getCurrentStream(state)));
  } else {
      bool succ =
        THCudaTensor_pointwiseApply2(state, dst, src, CopyOp<float>());
      THArgCheck(succ, 2, CUTORCH_DIM_WARNING);
  }

  if (srcDev != dstDev) {
    // synchronize remote device after copy
    hipEvent_t doneCopying;
    THCudaCheck(hipEventCreate(&doneCopying));
    THCudaCheck(hipEventRecord(
                  doneCopying,
                  THCState_getDeviceStream(state, copyDev, THCState_getCurrentStreamIndex(state))));
    THCudaCheck(hipSetDevice(remoteDev));
    THCudaCheck(hipStreamWaitEvent(
                  THCState_getDeviceStream(state, remoteDev, THCState_getCurrentStreamIndex(state)),
                  doneCopying, 0));
    THCudaCheck(hipEventDestroy(doneCopying));
  }

  if (curGPU() != oldDev) {
    THCudaCheck(hipSetDevice(oldDev));
  }

  hipError_t errcode = hipGetLastError();
  if (errcode != hipSuccess) {
    THError(hipGetErrorString(errcode));
  }
}
