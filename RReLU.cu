#include "hip/hip_runtime.h"
#include "THCApply.cuh"
#include "utils.h"
#include "common.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// copied from cutorch/lib/THC/THCTensorRandom.cu
#define MAX_NUM_BLOCKS 64
#define BLOCK_SIZE 256
#define NUM_BLOCKS(n) min((int)THCCeilDiv(n, (long) BLOCK_SIZE), MAX_NUM_BLOCKS)

__global__ void rreluUpdateOutputTrain(int n, hiprandStateMtgp32 *state, 
  float *input, float* noise, float *output, double a, double b)
{
  CUDA_KERNEL_LOOP(i, n)
  {
    if (input[i] <= 0)
    {
      float r = hiprand_uniform(&state[blockIdx.x]);
      r = r * (b-a) + a;
      output[i] = input[i] * r;
      noise[i] = r;
    }
    else
    {
      output[i] = input[i];
      noise[i] = 1;
    }
  }
}

struct RReLUUpdateOutputEval_functor
{
  const float negSlope_;

  RReLUUpdateOutputEval_functor(float negSlope) : negSlope_(negSlope) {}

  __device__ __forceinline__ void operator()(float* out, float* in)
  {
    const float x = *in;
    const float r = x <= 0 ? negSlope_ : 1;
    *out = x * r;
  }
};

struct RReLUUpdateOutputEvalIP_functor
{
  const float negSlope_;

  RReLUUpdateOutputEvalIP_functor(float negSlope) : negSlope_(negSlope) {}

  __device__ __forceinline__ void operator()(float* x)
  {
    if (*x <= 0)
    {
      *x = *x * negSlope_;
    }
  }
};

static int cunn_RReLU_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THCudaTensor *noise = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "noise", "torch.CudaTensor");
  double lower = luaT_getfieldchecknumber(L, 1, "lower");
  double upper = luaT_getfieldchecknumber(L, 1, "upper");
  int train = luaT_getfieldcheckboolean(L, 1, "train");
  int inplace = luaT_getfieldcheckboolean(L, 1, "inplace");

  THAssert(THCudaTensor_checkGPU(state, 3, input, output, noise));
  if (state->rngState->current_gen == NULL)
  {
    THError("Random number generators have not been initialized.");
  }

  if (train)
  {
    input = THCudaTensor_newContiguous(state, input);
    THCudaTensor_resizeAs(state, noise, input);
    float *input_data = THCudaTensor_data(state, input);
    float *noise_data = THCudaTensor_data(state, noise);
    long n = THCudaTensor_nElement(state, input);
    if (inplace)
    {
      rreluUpdateOutputTrain<<<NUM_BLOCKS(n), BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
        n, state->rngState->current_gen->gen_states, 
        input_data, noise_data, input_data, lower, upper);
      THCudaTensor_set(state, output, input);
    }
    else
    {
      THCudaTensor_resizeAs(state, output, input);
      float *output_data = THCudaTensor_data(state, output);
      rreluUpdateOutputTrain<<<NUM_BLOCKS(n), BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
        n, state->rngState->current_gen->gen_states, 
        input_data, noise_data, output_data, lower, upper);
    }
    THCudaTensor_free(state, input);
  }
  else
  {
    const double negSlope = (lower + upper) / 2;
    if (inplace)
    {
      THCudaTensor_pointwiseApply1(state, input, RReLUUpdateOutputEvalIP_functor(negSlope));
      THCudaTensor_set(state, output, input);
    }
    else
    {
      THCudaTensor_resizeAs(state, output, input);
      THCudaTensor_pointwiseApply2(state, output, input, RReLUUpdateOutputEval_functor(negSlope));
    }
  }

  return 1;
}

struct RReLUupdateGradInputEval_functor
{
  const float negSlope_;

  RReLUupdateGradInputEval_functor(float negSlope) : negSlope_(negSlope) {}

  __device__ __forceinline__ void operator()(float *gradIn, float *gradOut, float* in)
  {
    *gradIn = (*in) <= 0 ? (*gradOut) * negSlope_ : (*gradOut);
  }
};

struct RReLUupdateGradInputEvalIP_functor
{
  const float negSlope_;

  RReLUupdateGradInputEvalIP_functor(float negSlope) : negSlope_(negSlope) {}

  __device__ __forceinline__ void operator()(float *gradOut, float *in)
  {
    if (*in <= 0)
    {
      *gradOut = (*gradOut) * negSlope_;
    }
  }
};

static int cunn_RReLU_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  THCudaTensor *noise = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "noise", "torch.CudaTensor");
  double lower = luaT_getfieldchecknumber(L, 1, "lower");
  double upper = luaT_getfieldchecknumber(L, 1, "upper");
  int train = luaT_getfieldcheckboolean(L, 1, "train");
  int inplace = luaT_getfieldcheckboolean(L, 1, "inplace");
  
  THAssert(THCudaTensor_checkGPU(state, 4, input, gradOutput, gradInput, noise));
  
  gradOutput = THCudaTensor_newContiguous(state, gradOutput);
  
  if (train && upper - lower > 1E-6)    // e.g. if upper == lower, RReLU behaves like LeakyReLU
  {
    // multiply the gradient by the noise tensor
    if (inplace)
    {
      THCudaTensor_cmul(state, gradOutput, gradOutput, noise);
      THCudaTensor_set(state, gradInput, gradOutput);
    }
    else
    {
      THCudaTensor_resizeAs(state, gradInput, input);
      THCudaTensor_cmul(state, gradInput, gradOutput, noise);
    }    
  }
  else
  {
    // use constant factor for negative input values
    const double negSlope = (lower + upper) / 2;
    if (inplace)
    {
      THCudaTensor_pointwiseApply2(state, gradOutput, input, RReLUupdateGradInputEvalIP_functor(negSlope));
      THCudaTensor_set(state, gradInput, gradOutput);
    }
    else
    {
      THCudaTensor_resizeAs(state, gradInput, input);
      THCudaTensor_pointwiseApply3(state, gradInput, gradOutput, input, RReLUupdateGradInputEval_functor(negSlope));
    }
  }
  
  THCudaTensor_free(state, gradOutput);
  return 1;
}

static const struct luaL_Reg cunn_RReLU__ [] = {
  {"RReLU_updateOutput", cunn_RReLU_updateOutput},
  {"RReLU_updateGradInput", cunn_RReLU_updateGradInput},
  {NULL, NULL}
};

void cunn_RReLU_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_RReLU__, "nn");
  lua_pop(L,1);
}
