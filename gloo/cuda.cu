#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2017-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree. An additional grant
 * of patent rights can be found in the PATENTS file in the same directory.
 */

#include "gloo/hip/hip_runtime.h"
#include "gloo/cuda_private.h"

namespace gloo {

const hipStream_t kStreamNotSet = (hipStream_t)(-1);

// Default mutex to synchronize contentious CUDA and NCCL operations
static std::mutex defaultCudaMutex;
std::atomic<std::mutex*> CudaShared::mutex_(&defaultCudaMutex);

CudaStream::CudaStream(int deviceId, hipStream_t stream)
    : deviceId_(deviceId),
      stream_(stream),
      streamOwner_(false) {
  CudaDeviceScope scope(deviceId_);

  // Create new stream if it wasn't specified
  if (stream_ == kStreamNotSet) {
    int loPri, hiPri;
    CUDA_CHECK(hipDeviceGetStreamPriorityRange(&loPri, &hiPri));
    CUDA_CHECK(hipStreamCreateWithPriority(
                 &stream_, hipStreamNonBlocking, hiPri));
    streamOwner_ = true;
  }

  // Create new event to synchronize operations against
  CUDA_CHECK(hipEventCreateWithFlags(&event_, hipEventDisableTiming));
}

CudaStream::CudaStream(CudaStream&& other) noexcept
    : deviceId_(other.deviceId_),
      stream_(other.stream_),
      streamOwner_(other.streamOwner_),
      event_(other.event_) {
  other.stream_ = nullptr;
  other.event_ = nullptr;
}

CudaStream::~CudaStream() {
  if (event_ != nullptr) {
    // Make sure outstanding operations are complete. If the event
    // hasn't been queued this call will return immediately.
    CUDA_CHECK(hipEventSynchronize(event_));
    CUDA_CHECK(hipEventDestroy(event_));
  }
  if (streamOwner_ && stream_ != nullptr) {
    CUDA_CHECK(hipStreamDestroy(stream_));
  }
}

template <typename T>
void CudaStream::copyAsync(
    CudaHostPointer<T>& dst,
    CudaDevicePointer<T>& src) {
  CudaDeviceScope scope(deviceId_);
  GLOO_ENFORCE_LE(dst.getCount(), src.getCount());
  CUDA_CHECK(hipMemcpyAsync(
                 *dst,
                 *src,
                 dst.getCount() * sizeof(T),
                 hipMemcpyDeviceToHost,
                 stream_));
  CUDA_CHECK(hipEventRecord(event_, stream_));
}

template <typename T>
void CudaStream::copyAsync(
    CudaHostPointer<T>& dst,
    CudaHostPointer<T>& src) {
  CudaDeviceScope scope(deviceId_);
  GLOO_ENFORCE_LE(dst.getCount(), src.getCount());
  CUDA_CHECK(hipMemcpyAsync(
                 *dst,
                 *src,
                 dst.getCount() * sizeof(T),
                 hipMemcpyHostToHost,
                 stream_));
  CUDA_CHECK(hipEventRecord(event_, stream_));
}

template <typename T>
void CudaStream::copyAsync(
    CudaDevicePointer<T>& dst,
    CudaDevicePointer<T>& src) {
  CudaDeviceScope scope(deviceId_);
  GLOO_ENFORCE_LE(dst.getCount(), src.getCount());
  CUDA_CHECK(hipMemcpyAsync(
                 *dst,
                 *src,
                 dst.getCount() * sizeof(T),
                 hipMemcpyDeviceToDevice,
                 stream_));
  CUDA_CHECK(hipEventRecord(event_, stream_));
}

template <typename T>
void CudaStream::copyAsync(
    CudaDevicePointer<T>& dst,
    CudaHostPointer<T>& src) {
  CudaDeviceScope scope(deviceId_);
  GLOO_ENFORCE_LE(dst.getCount(), src.getCount());
  CUDA_CHECK(hipMemcpyAsync(
                 *dst,
                 *src,
                 dst.getCount() * sizeof(T),
                 hipMemcpyHostToDevice,
                 stream_));
  CUDA_CHECK(hipEventRecord(event_, stream_));
}

void CudaStream::record() {
  CUDA_CHECK(hipEventRecord(event_, stream_));
}

void CudaStream::wait() {
  CudaDeviceScope scope(deviceId_);
  CUDA_CHECK(hipEventSynchronize(event_));
}

template <typename T>
CudaDevicePointer<T> CudaDevicePointer<T>::alloc(
    size_t count) {
  T* ptr = nullptr;
  size_t bytes = count * sizeof(T);
  {
    std::lock_guard<std::mutex> lock(CudaShared::getMutex());
    CUDA_CHECK(hipMalloc(&ptr, bytes));
  }
  auto p = create(ptr, count);
  p.owner_ = true;
  return p;
}

template<typename T>
CudaDevicePointer<T> CudaDevicePointer<T>::create(
    T* ptr,
    size_t count) {
  CudaDevicePointer p(ptr, count, false);
  return p;
}

template<typename T>
CudaDevicePointer<T>::CudaDevicePointer(T* ptr, size_t count, bool owner)
    : device_(ptr),
      count_(count),
      owner_(owner),
      deviceId_(getGPUIDForPointer(device_)) {
}

template<typename T>
CudaDevicePointer<T>::CudaDevicePointer(CudaDevicePointer<T>&& other) noexcept
    : device_(other.device_),
      count_(other.count_),
      owner_(other.owner_),
      deviceId_(other.deviceId_) {
  // Nullify fields that would otherwise be destructed
  other.device_ = nullptr;
  other.owner_ = false;
}

template<typename T>
CudaDevicePointer<T>& CudaDevicePointer<T>::operator=(
    CudaDevicePointer<T>&& other) {
  device_ = other.device_;
  count_ = other.count_;
  owner_ = other.owner_;
  deviceId_ = other.deviceId_;

  // Nullify fields that would otherwise be destructed
  other.device_ = nullptr;
  other.owner_ = false;

  return *this;
}

template<typename T>
CudaDevicePointer<T>::~CudaDevicePointer() {
  if (deviceId_ < 0) {
    return;
  }
  CudaDeviceScope scope(deviceId_);
  if (owner_ && device_ != nullptr) {
    std::lock_guard<std::mutex> lock(CudaShared::getMutex());
    CUDA_CHECK(hipFree(device_));
  }
}

template <typename T>
CudaHostPointer<T> CudaHostPointer<T>::alloc(size_t count) {
  T* ptr = nullptr;
  size_t bytes = count * sizeof(T);
  {
    std::lock_guard<std::mutex> lock(CudaShared::getMutex());
    CUDA_CHECK(hipHostMalloc(&ptr, bytes));
  }
  return CudaHostPointer<T>(ptr, count, true);
}

template <typename T>
CudaHostPointer<T>::CudaHostPointer(T* ptr, size_t count, bool owner)
    : host_(ptr),
      count_(count),
      owner_(owner) {}

template <typename T>
CudaHostPointer<T>::CudaHostPointer(CudaHostPointer&& other) noexcept
    : host_(other.host_),
      count_(other.count_),
      owner_(other.owner_) {
  other.host_ = nullptr;
  other.count_ = 0;
  other.owner_ = false;
}

template<typename T>
CudaHostPointer<T>& CudaHostPointer<T>::operator=(CudaHostPointer<T>&& other) {
  host_ = other.host_;
  count_ = other.count_;
  owner_ = other.owner_;
  other.host_ = nullptr;
  other.count_ = 0;
  other.owner_ = false;
  return *this;
}

template<typename T>
CudaHostPointer<T>::~CudaHostPointer() {
  if (owner_) {
    std::lock_guard<std::mutex> lock(CudaShared::getMutex());
    CUDA_CHECK(hipHostFree(host_));
  }
}

// Instantiate templates
template class CudaDevicePointer<float>;
template class CudaHostPointer<float>;

template void CudaStream::copyAsync<float>(
    CudaHostPointer<float>& dst,
    CudaDevicePointer<float>& src);

template void CudaStream::copyAsync<float>(
    CudaHostPointer<float>& dst,
    CudaHostPointer<float>& src);

template void CudaStream::copyAsync<float>(
    CudaDevicePointer<float>& dst,
    CudaDevicePointer<float>& src);

template void CudaStream::copyAsync<float>(
    CudaDevicePointer<float>& dst,
    CudaHostPointer<float>& src);

// Borrowed limits from Caffe2 code (see core/common_gpu.h)
constexpr static int kCudaNumThreads = 512;
constexpr static int kCudaMaximumNumBlocks = 4096;

static inline int cudaGetBlocks(const int N) {
  return std::min((N + kCudaNumThreads - 1) / kCudaNumThreads,
                  kCudaMaximumNumBlocks);
}

#define DELEGATE_SIMPLE_CUDA_BINARY_OPERATOR(T, Funcname, op)           \
  __global__                                                            \
  void _Kernel_##T##_##Funcname(T* dst, const T* src, const int n) {    \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;                 \
         i < (n);                                                       \
         i += blockDim.x * gridDim.x) {                                 \
      dst[i] = dst[i] op src[i];                                        \
    }                                                                   \
  }                                                                     \
  template <>                                                           \
  void Funcname<T>(                                                     \
    T* dst,                                                             \
    const T* src,                                                       \
    size_t n,                                                           \
    const hipStream_t stream) {                                        \
    _Kernel_##T##_##Funcname<<<                                         \
      cudaGetBlocks(n),                                                 \
      kCudaNumThreads,                                                  \
      0,                                                                \
      stream>>>(                                                        \
        dst, src, n);                                                   \
  }

DELEGATE_SIMPLE_CUDA_BINARY_OPERATOR(float, cudaSum, +);
DELEGATE_SIMPLE_CUDA_BINARY_OPERATOR(float, cudaProduct, *);

#define DELEGATE_SIMPLE_CUDA_BINARY_COMPARE(T, Funcname, op)            \
  __global__                                                            \
  void _Kernel_##T##_##Funcname(T* dst, const T* src, const int n) {    \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;                 \
         i < (n);                                                       \
         i += blockDim.x * gridDim.x) {                                 \
      if (src[i] op dst[i]) {                                           \
        dst[i] = src[i];                                                \
      }                                                                 \
    }                                                                   \
  }                                                                     \
  template <>                                                           \
  void Funcname<T>(                                                     \
    T* dst,                                                             \
    const T* src,                                                       \
    size_t n,                                                           \
    const hipStream_t stream) {                                        \
    _Kernel_##T##_##Funcname<<<                                         \
      cudaGetBlocks(n),                                                 \
      kCudaNumThreads,                                                  \
      0,                                                                \
      stream>>>(                                                        \
        dst, src, n);                                                   \
  }

DELEGATE_SIMPLE_CUDA_BINARY_COMPARE(float, cudaMin, <);
DELEGATE_SIMPLE_CUDA_BINARY_COMPARE(float, cudaMax, >);

} // namespace gloo
