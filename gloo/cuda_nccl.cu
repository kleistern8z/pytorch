/**
 * Copyright (c) 2017-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree. An additional grant
 * of patent rights can be found in the PATENTS file in the same directory.
 */

#include "gloo/cuda_nccl.h"

#include "gloo/cuda_private.h"

namespace gloo {
namespace nccl {

NCCLContext::NCCLContext(
    int device,
    hipStream_t stream,
    std::vector<NCCLElement>&& elements,
    int root)
    : masterDevice(device),
      masterStream(stream),
      root(root),
      elements(elements) {
  std::vector<int> devices;
  devices.reserve(elements.size());
  for (auto el : elements) {
    devices.push_back(el.device);
  }
  {
    // Initialze comms. Synchronize with conflicting CUDA and NCCL operations.
    std::lock_guard<std::mutex> lock(CudaShared::getMutex());
    comms.resize(elements.size());
    NCCL_CHECK(ncclCommInitAll(comms.data(), devices.size(), devices.data()));
  }
  // Allocate the events and streams
  events.resize(elements.size());
  for (auto i = 0; i < elements.size(); i++) {
    CudaDeviceScope scope(elements[i].device);
    CUDA_CHECK(hipEventCreateWithFlags(
        &events[i], hipEventDefault | hipEventDisableTiming));
  }
  CUDA_CHECK(hipEventCreateWithFlags(
      &masterEvent, hipEventDefault | hipEventDisableTiming));
}

NCCLContext::NCCLContext(NCCLContext&& other) noexcept
  : masterDevice(other.masterDevice),
    masterEvent(other.masterEvent),
    masterStream(other.masterStream),
    root(other.root),
    elements(std::move(other.elements)),
    comms(std::move(other.comms)),
    events(std::move(other.events)) {
  // Nullify fields that would otherwise be destructed
  other.masterEvent = nullptr;
}

NCCLContext::~NCCLContext() {
  if (masterEvent != nullptr) {
    CudaDeviceScope scope(masterDevice);
    // Make sure outstanding operations are complete. If the event
    // hasn't been queued this call will return immediately.
    CUDA_CHECK(hipEventSynchronize(masterEvent));
    CUDA_CHECK(hipEventDestroy(masterEvent));
  }
  for (auto i = 0; i < elements.size(); ++i) {
    CudaDeviceScope scope(elements[i].device);
    CUDA_CHECK(hipEventDestroy(events[i]));
    {
      // Synchronize memory allocation with NCCL operations
      std::lock_guard<std::mutex> lock(CudaShared::getMutex());
      ncclCommDestroy(comms[i]);
    }
  }
}

template <typename T>
class ncclTypeWrapper;

template <>
class ncclTypeWrapper<float> {
 public:
  static const ncclDataType_t type = ncclFloat;
};

template <typename T>
void NCCLOp<T>::wait() {
  CudaDeviceScope scope(context_.masterDevice);
  CUDA_CHECK(hipEventSynchronize(context_.masterEvent));
}

template <typename T>
template <typename F>
void NCCLOp<T>::runNCCL(F&& f) {
  // Record an event on the master stream and wait on it in each of the child
  // streams to ensure both are synchronized.
  {
    CudaDeviceScope scope(context_.masterDevice);
    CUDA_CHECK(
        hipEventRecord(context_.masterEvent, context_.masterStream));
  }

  // Kick off the NCCL operation on each device
  {
    // Synchronize memory allocation with NCCL operations
    std::lock_guard<std::mutex> lock(CudaShared::getMutex());

    const auto& elements = context_.elements;
    for (auto i = 0; i < elements.size(); i++) {
      const auto& element = elements[i];
      const auto& comm = context_.comms[i];
      const auto& event = context_.events[i];
      const auto& stream = element.stream;
      // Synchronize with the master stream
      CudaDeviceScope scope(element.device);
      CUDA_CHECK(hipStreamWaitEvent(stream, context_.masterEvent, 0));
      // Run the operation
      f(element, comm, stream);
      CUDA_CHECK(hipEventRecord(event, stream));
    }
  }

  // Synchronize with the master stream.
  CudaDeviceScope scope(context_.masterDevice);
  for (auto& event : context_.events) {
    CUDA_CHECK(hipStreamWaitEvent(context_.masterStream, event, 0));
  }
  // Record an event on the master stream to signal end of the operation.
  CUDA_CHECK(
      hipEventRecord(context_.masterEvent, context_.masterStream));
}

template <typename T>
void ReduceOp<T>::runAsync() {
  const auto root = this->context_.root;
  this->runNCCL([root](
      const NCCLElement& element, ncclComm_t comm, hipStream_t stream) {
    NCCL_CHECK(ncclReduce(
        element.src,
        element.dst,
        element.length,
        ncclTypeWrapper<T>::type,
        ncclSum,
        root,
        comm,
        stream));
  });
}

template <typename T>
void BroadcastOp<T>::runAsync() {
  const auto root = this->context_.root;
  this->runNCCL([root](
      const NCCLElement& element, ncclComm_t comm, hipStream_t stream) {
    NCCL_CHECK(ncclBcast(
        element.dst,
        element.length,
        ncclTypeWrapper<T>::type,
        root,
        comm,
        stream));
  });
}

template class NCCLOp<float>;
template class ReduceOp<float>;
template class BroadcastOp<float>;

} // namespace nccl
} // namespace gloo
