#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2017-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree. An additional grant
 * of patent rights can be found in the PATENTS file in the same directory.
 */

#include "gloo/cuda_private.h"

#include "gloo/common/common.h"

namespace gloo {

std::mutex gCudaMutex;

template<typename T>
__global__ void initializeMemory(T* ptr, const T val, const size_t n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  for (; i < n; i += blockDim.x) {
    ptr[i] = val;
  }
}

template<typename T>
CudaMemory<T>::CudaMemory(size_t n): n_(n), bytes_(n * sizeof(T)) {
  CUDA_CHECK(hipGetDevice(&device_));
  // Sychronize memory allocation with NCCL operations
  std::lock_guard<std::mutex> lock(gCudaMutex);
  CUDA_CHECK(hipMalloc(&ptr_, bytes_));
}

template<typename T>
CudaMemory<T>::CudaMemory(CudaMemory<T>&& other) noexcept
  : n_(other.n_),
    bytes_(other.bytes_),
    device_(other.device_),
    ptr_(other.ptr_) {
  // Nullify pointer on move source
  other.ptr_ = nullptr;
}

template<typename T>
CudaMemory<T>::~CudaMemory() {
  CudaDeviceScope scope(device_);
  if (ptr_ != nullptr) {
    // Sychronize memory allocation with NCCL operations
    std::lock_guard<std::mutex> lock(gCudaMutex);
    CUDA_CHECK(hipFree(ptr_));
  }
}

template<typename T>
void CudaMemory<T>::set(T val, hipStream_t stream) {
  CudaDeviceScope scope(device_);
  if (stream == kStreamNotSet) {
    initializeMemory<<<1, 32>>>(ptr_, val, n_);
  } else {
    initializeMemory<<<1, 32, 0, stream>>>(ptr_, val, n_);
  }
}

template<typename T>
std::unique_ptr<T[]> CudaMemory<T>::copyToHost() {
  auto host = make_unique<T[]>(n_);
  hipMemcpy(host.get(), ptr_, bytes_, hipMemcpyDefault);
  return host;
}

// Instantiate template
template class CudaMemory<float>;

} // namespace gloo
