#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "THCAtomics.cuh"

#define CUDA_MAX_THREADS 1024   // this is safe, in reality 256 is our limit

/*
 * Description:
 *    this function adaptively maxpools an input 4D tensor along dimensions 2 and 3
 *    4D input, 4D output, 4D argmax x and y
 */
 template <typename T>
__global__ void adaptivemaxpool(T *input, T *output, THCIndex_t *indices,
                        int input_n, int input_h, int input_w,
                        int output_h, int output_w,
                        int strideh, int stridew,
                        int strided)
{
  // iterators
  int xx, yy;

  // compute offsets based on thread/block ID
  int o = blockIdx.x;
  int i = o;
  //int k = blockIdx.x % input_n;

  int xx_start = threadIdx.x;
  int xx_end = output_w;
  const int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  const int yy_step = blockDim.y*gridDim.y;
  // select input/output plane
  output = output + o*output_w*output_h;
  input = input + i*strided;
  indices = indices + o*output_w*output_h;

  // For all output pixels...
  for(yy = yy_start; yy < yy_end; yy+=yy_step) {

    int y_start = (int)floor(float(yy) / output_h * input_h);
    int y_end   = (int)ceil(float(yy+1) / output_h * input_h);
    int kH = y_end-y_start;

    for(xx = xx_start; xx < xx_end; xx+=xx_step) {
      int x_start = (int)floor(float(xx) / output_w * input_w);
      int x_end   = (int)ceil(float(xx + 1) / output_w * input_w);

      int kW = x_end-x_start;

      // Compute the mean of the input image...
      T *ptr_input = input + y_start*strideh + x_start*stridew;
      T *ptr_output = output + yy*output_w + xx;
      THCIndex_t *ptr_ind = indices + yy*output_w + xx;
      int argmax = -1;
      T max = THCNumerics<T>::min();
      int kx, ky;
      for(ky = 0; ky < kH; ky++) {
        for(kx = 0; kx < kW; kx++) {
          T val = ptr_input[kx*stridew];
          if (val > max) {
            max = val;
            argmax = (ky+y_start)*input_w + kx+x_start;
          }
        }
        ptr_input += strideh; // next input line
      }
      // Update output and argmax
      *ptr_output = max;
      *ptr_ind = argmax + TH_INDEX_BASE;
    }
  }
}

/*
 * Description:
 *    this function computes the gradInput from weight and gradOutput
 */
 template <typename T>
__global__ void adaptivemaxgradinput(T *gradInput, T *gradOutput, THCIndex_t *indices,
                             int input_n, int input_h, int input_w,
                             int output_h, int output_w)
{
  // iterators
  int xx, yy;

  // compute offsets based on thread/block ID
  int o = blockIdx.x;
  int i = o;
  //int k = blockIdx.x % input_n;

  int xx_start = threadIdx.x;
  int xx_end = output_w;
  int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  int yy_step = blockDim.y*gridDim.y;

  // select input/output plane
  gradOutput = gradOutput + o*output_w*output_h;
  gradInput = gradInput + i*input_w*input_h;
  indices = indices + o*output_w*output_h;

  // compute gradInput
  for(yy = yy_start; yy < yy_end; yy+=yy_step) {

    int y_start = (int)floor(float(yy) / output_h * input_h);

    for(xx = xx_start; xx < xx_end; xx+=xx_step) {

      int x_start = (int)floor(float(xx) / output_w * input_w);

      T *ptr_gradInput = gradInput + y_start*input_w + x_start;
      T *ptr_gradOutput = gradOutput + yy*output_w + xx;
      THCIndex_t *ptr_ind = indices + yy*output_w + xx;
      T z = *ptr_gradOutput;

      int argmax = (*ptr_ind) - TH_INDEX_BASE - x_start - y_start*input_w;

      ptr_gradInput[argmax] += z;
    }
  }
}

/*
 * Description:
 *    this function computes the gradInput from weight and gradOutput
 *    when kH != dH or kW != dW (uses atomic add)
 */
 template <typename T>
__global__ void atomicadaptivemaxgradinput(
  T *gradInput, T *gradOutput, THCIndex_t *indices,
  int input_n, int input_h, int input_w, int output_h, int output_w
)
{
  // iterators
  int xx, yy;

  // compute offsets based on thread/block ID
  int o = blockIdx.x;
  int i = o;

  int xx_start = threadIdx.x;
  int xx_end = output_w;
  int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  int yy_step = blockDim.y*gridDim.y;

  // select input/output plane
  gradOutput = gradOutput + o*output_w*output_h;
  gradInput = gradInput + i*input_w*input_h;
  indices = indices + o*output_w*output_h;

  // compute gradInput
  for(yy = yy_start; yy < yy_end; yy+=yy_step) {

    int y_start = (int)floor(float(yy) / output_h * input_h);

    for(xx = xx_start; xx < xx_end; xx+=xx_step) {

      int x_start = (int)floor(float(xx) / output_w * input_w);

      T *ptr_gradInput = gradInput + y_start*input_w + x_start;
      T *ptr_gradOutput = gradOutput + yy*output_w + xx;
      THCIndex_t *ptr_ind = indices + yy*output_w + xx;
      T z = *ptr_gradOutput;

      int argmax = (*ptr_ind) - TH_INDEX_BASE - x_start - y_start*input_w;

      // atomic add since different threads could update same variable
      atomicAdd(&(ptr_gradInput[argmax]), z);
    }
  }
}

#include "generic/SpatialAdaptiveMaxPooling.cu"
#include "THCGenerateFloatTypes.h"

#undef CUDA_MAX_THREADS
