#include "THCStorage.h"

#include <thrust/device_ptr.h>
#include <thrust/fill.h>

void THCudaStorage_fill(THCState *state, THCudaStorage *self, float value)
{
  thrust::device_ptr<float> self_data(self->data);
  thrust::fill(self_data, self_data+self->size, value);
}

void THCudaStorage_resize(THCState *state, THCudaStorage *self, long size)
{
  THArgCheck(size >= 0, 2, "invalid size");

  if(!(self->flag & TH_STORAGE_RESIZABLE))
    return;

  if(size == 0)
  {
    if(self->flag & TH_STORAGE_FREEMEM)
      THCudaCheck(hipFree(self->data));
    self->data = NULL;
    self->size = 0;
  }
  else
  {
    float *data;
    THCudaCheck(hipMalloc((void**)(&data), size * sizeof(float)));
    THCudaCheck(hipMemcpyAsync(data, self->data, THMin(self->size, size) * sizeof(float), hipMemcpyDeviceToDevice));
    THCudaCheck(hipFree(self->data));
    self->data = data;
    self->size = size;
  }
}
