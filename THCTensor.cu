#include "THCTensor.h"

hipTextureObject_t THCudaTensor_getTextureObject(THCudaTensor *self)
{
  hipTextureObject_t texObj;
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeLinear;
  resDesc.res.linear.devPtr = THCudaTensor_data(self);
  resDesc.res.linear.sizeInBytes = THCudaTensor_nElement(self) * 4;
  resDesc.res.linear.desc = hipCreateChannelDesc(32, 0, 0, 0, 
                                                  hipChannelFormatKindFloat);
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
  hipError_t errcode = hipGetLastError();
  if(errcode != hipSuccess) {
    if (THCudaTensor_nElement(self) > 2>>27)
      THError("Failed to create texture object, "
              "nElement:%ld exceeds 27-bit addressing required for tex1Dfetch. Cuda Error: %s", 
              THCudaTensor_nElement(self), hipGetErrorString(errcode));
    else
      THError("Failed to create texture object: %s", hipGetErrorString(errcode));
  }
  return texObj;
}
