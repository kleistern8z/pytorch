#include "THCTensor.h"

hipTextureObject_t THCudaTensor_getTextureObject(THCudaTensor *self)
{
  hipTextureObject_t texObj;
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeLinear;
  resDesc.res.linear.devPtr = THCudaTensor_data(self);
  resDesc.res.linear.sizeInBytes = THCudaTensor_nElement(self) * 4;
  resDesc.res.linear.desc = hipCreateChannelDesc(32, 0, 0, 0, 
                                                  hipChannelFormatKindFloat);
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  checkCudaErrors(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL));
  return texObj;
}
