#include "hip/hip_runtime.h"
#include "utils.h"

#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>
#if CUDA_VERSION >= 7000
#include <thrust/system/cuda/execution_policy.h>
#endif

struct smoothl1_functor
{
  smoothl1_functor() {}

  __host__ __device__ float operator()(const float& x, const float& y) const
    {
      float z = fabsf(x-y);
      return z < 1.f ? 0.5f*z*z : z - 0.5f;
  }
};


static int cunn_SmoothL1Criterion_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *target = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  THAssert(THCudaTensor_checkGPU(state, 2, input, target));

  int sizeAverage = luaT_getfieldcheckboolean(L, 1, "sizeAverage");
  luaL_argcheck(L, THCudaTensor_nElement(state, input) == THCudaTensor_nElement(state, target), 2,
                "input and target need to have the same number of elements");

  float sum;

  long size = THCudaTensor_nElement(state, input);

  input = THCudaTensor_newContiguous(state, input);
  target = THCudaTensor_newContiguous(state, target);

  thrust::device_ptr<float> input_data(THCudaTensor_data(state, input));
  thrust::device_ptr<float> target_data(THCudaTensor_data(state, target));
  sum = thrust::inner_product(
#if CUDA_VERSION >= 7000
    thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
    input_data, input_data+size, target_data, (float) 0,
    thrust::plus<float>(), smoothl1_functor());

  if(sizeAverage)
    sum /= size;

  THCudaTensor_free(state, input);
  THCudaTensor_free(state, target);

  lua_pushnumber(L, sum);
  lua_setfield(L, 1, "output");

  lua_pushnumber(L, sum);
  return 1;
}


struct smoothl1_updateGradInput_functor
{
  const float norm;

  smoothl1_updateGradInput_functor(float norm_) : norm(norm_) {}

  __host__ __device__ float operator()(const float& x, const float& y) const
    {
      float z = x - y;
      if(z < -1.f)
        return -norm;
      else if(z > 1.f)
        return norm;
      else
        return norm * z;
  }
};

static int cunn_SmoothL1Criterion_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *target = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  int sizeAverage = luaT_getfieldcheckboolean(L, 1, "sizeAverage");
  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  luaL_argcheck(L, THCudaTensor_nElement(state, input) == THCudaTensor_nElement(state, target), 2,
                "input and target need to have the same number of elements");
  THAssert(THCudaTensor_checkGPU(state, 3, input, target, gradInput));

  long size = THCudaTensor_nElement(state, input);
  float norm = (sizeAverage ? 1./size : 1.);

  input = THCudaTensor_newContiguous(state, input);
  target = THCudaTensor_newContiguous(state, target);

  THCudaTensor_resizeAs(state, gradInput, input);

  thrust::device_ptr<float> input_data(THCudaTensor_data(state, input));
  thrust::device_ptr<float> target_data(THCudaTensor_data(state, target));
  thrust::device_ptr<float> gradInput_data(THCudaTensor_data(state, gradInput));

  thrust::transform(
#if CUDA_VERSION >= 7000
    thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
    input_data, input_data+size, target_data, gradInput_data,
    smoothl1_updateGradInput_functor(norm));

  THCudaTensor_free(state, input);
  THCudaTensor_free(state, target);
  return 1;
}

static const struct luaL_Reg cunn_SmoothL1Criterion__ [] = {
  {"SmoothL1Criterion_updateOutput", cunn_SmoothL1Criterion_updateOutput},
  {"SmoothL1Criterion_updateGradInput", cunn_SmoothL1Criterion_updateGradInput},
  {NULL, NULL}
};

void cunn_SmoothL1Criterion_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_SmoothL1Criterion__, "nn");
  lua_pop(L,1);
}
