#include "hip/hip_runtime.h"
#include "THCTensorRandom.h"
#include "THCDeviceUtils.cuh"
#include "THCGeneral.h"
#include "THCTensorCopy.h"
#include "THCTensorMath.h"
#include "THCReduceApplyUtils.cuh"
#include "THCTensorRandom.cuh"

#include <thrust/functional.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_mtgp32_host.h>
#include <rocrand/rocrand_mtgp32_11213.h>

#define MAX_NUM_BLOCKS 64
#define BLOCK_SIZE 256

/* Sets up generator. Allocates but does not create the generator states. */
__host__ void initializeGenerator(THCState *state, Generator* gen)
{
  THCudaCheck(THCudaMalloc(state, (void**)&gen->gen_states, MAX_NUM_BLOCKS * sizeof(hiprandStateMtgp32)));
  THCudaCheck(THCudaMalloc(state, (void**)&gen->kernel_params, sizeof(mtgp32_kernel_params)));
}

/* Frees memory allocated during setup. */
__host__ void destroyGenerator(THCState *state, Generator* gen)
{
  if (gen->gen_states)
  {
    THCudaCheck(THCudaFree(state, gen->gen_states));
    gen->gen_states = NULL;
  }
  if (gen->kernel_params)
  {
    THCudaCheck(THCudaFree(state, gen->kernel_params));
    gen->kernel_params = NULL;
  }
}

/* Creates a new generator state given the seed. */
__host__ void createGeneratorState(Generator* gen, unsigned long seed)
{
  if (hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, gen->kernel_params) != HIPRAND_STATUS_SUCCESS)
  {
    THError("Creating MTGP constants failed.");
  }
  if (hiprandMakeMTGP32KernelState(gen->gen_states, mtgp32dc_params_fast_11213,
                                  gen->kernel_params, MAX_NUM_BLOCKS, seed) != HIPRAND_STATUS_SUCCESS)
  {
    THError("Creating MTGP kernel state failed.");
  }
}

/* Initialize generator array (must be called before any other function) */
__host__ void THCRandom_init(THCState* state, int devices, int current_device)
{
  THCRNGState* rng_state = THCState_getRngState(state);
  rng_state->num_devices = devices;
  rng_state->gen = (Generator*)malloc(rng_state->num_devices * sizeof(Generator));
  for (int i = 0; i < rng_state->num_devices; ++i)
  {
    rng_state->gen[i].initf = 0;
    rng_state->gen[i].initial_seed = 0;
    rng_state->gen[i].gen_states = NULL;
    rng_state->gen[i].kernel_params = NULL;
  }
}

/* Destroy generators and free memory */
__host__ void THCRandom_shutdown(THCState* state)
{
  THCRNGState* rng_state = THCState_getRngState(state);
  if (rng_state->gen == NULL) return;
  for (int i = 0; i < rng_state->num_devices; ++i)
  {
    destroyGenerator(state, &rng_state->gen[i]);
  }
  free(rng_state->gen);
  rng_state->gen = NULL;
}

/* Manually set the generator seed */
__host__ static void THCRandom_manualSeedGen(Generator* gen, unsigned long seed)
{
  gen->initial_seed = seed;
  createGeneratorState(gen, seed);
  gen->initf = 1;
}

/* Get the generator for the current device */
__host__ Generator* THCRandom_getGenerator(THCState* state)
{
  THCRNGState* rng_state = THCState_getRngState(state);

  int device;
  THCudaCheck(hipGetDevice(&device));
  if (device >= rng_state->num_devices) THError("Invalid device index.");

  Generator* gen = &rng_state->gen[device];
  if (gen->initf == 0)
  {
    initializeGenerator(state, gen);
    THCRandom_manualSeedGen(gen, (unsigned long)time(0));
  }
  return gen;
}

__host__ struct hiprandStateMtgp32* THCRandom_generatorStates(struct THCState* state)
{
  return THCRandom_getGenerator(state)->gen_states;
}

/* Random seed */
__host__ unsigned long THCRandom_seed(THCState* state)
{
  unsigned long s = (unsigned long)time(0);
  THCRandom_manualSeed(state, s);
  return s;
}

__host__ unsigned long THCRandom_seedAll(THCState* state)
{
  unsigned long s = (unsigned long)time(0);
  THCRandom_manualSeedAll(state, s);
  return s;
}

/* Manually set the seed */
__host__ void THCRandom_manualSeed(THCState* state, unsigned long seed)
{
  Generator* gen = THCRandom_getGenerator(state);
  THCRandom_manualSeedGen(gen, seed);
}

__host__ void THCRandom_manualSeedAll(THCState* state, unsigned long seed)
{
  THCRNGState* rng_state = THCState_getRngState(state);
  int currentDevice;
  THCudaCheck(hipGetDevice(&currentDevice));
  for (int i = 0; i < rng_state->num_devices; ++i) {
    THCudaCheck(hipSetDevice(i));
    THCRandom_manualSeed(state, seed);
  }
  THCudaCheck(hipSetDevice(currentDevice));
}

/* Get the initial seed */
__host__ unsigned long THCRandom_initialSeed(THCState* state)
{
  return THCRandom_getGenerator(state)->initial_seed;
}

__host__ void THCRandom_getRNGState(THCState* state, THByteTensor *rng_state)
{
  Generator* gen = THCRandom_getGenerator(state);

  // The RNG state comprises the MTPG32 states and the seed.
  static const size_t states_size = MAX_NUM_BLOCKS * sizeof(hiprandStateMtgp32);
  static const size_t seed_size = sizeof(unsigned long);
  static const size_t total_size = states_size + seed_size;
  THByteTensor_resize1d(rng_state, total_size);
  THArgCheck(THByteTensor_nElement(rng_state) == total_size, 1, "RNG state is wrong size");
  THArgCheck(THByteTensor_isContiguous(rng_state), 1, "RNG state must be contiguous");
  THCudaCheck(hipMemcpy(THByteTensor_data(rng_state), gen->gen_states,
                         states_size, hipMemcpyDeviceToHost));
  memcpy(THByteTensor_data(rng_state) + states_size, &gen->initial_seed, seed_size);
}

__global__ void set_rngstate_kernel(hiprandStateMtgp32 *state, mtgp32_kernel_params *kernel)
{
  state[threadIdx.x].k = kernel;
}

__host__ void THCRandom_setRNGState(THCState* state, THByteTensor *rng_state)
{
  Generator* gen = THCRandom_getGenerator(state);

  static const size_t states_size = MAX_NUM_BLOCKS * sizeof(hiprandStateMtgp32);
  static const size_t seed_size = sizeof(unsigned long);
  static const size_t total_size = states_size + seed_size;
  THArgCheck(THByteTensor_nElement(rng_state) == total_size, 1, "RNG state is wrong size");
  THArgCheck(THByteTensor_isContiguous(rng_state), 1, "RNG state must be contiguous");

  THCudaCheck(hipMemcpy(gen->gen_states, THByteTensor_data(rng_state),
                         states_size, hipMemcpyHostToDevice));
  set_rngstate_kernel<<<1, MAX_NUM_BLOCKS, 0, THCState_getCurrentStream(state)>>>(
      gen->gen_states, gen->kernel_params);
  memcpy(&gen->initial_seed, THByteTensor_data(rng_state) + states_size, seed_size);
}

#define GENERATE_KERNEL1(NAME, T, ARG1, CURAND_T, CURAND_FUNC, TRANSFORM)               \
__global__ void NAME(hiprandStateMtgp32 *state, int size, T *result, ARG1)  \
{                                                                              \
  int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;                             \
  int rounded_size = THCCeilDiv(size, BLOCK_SIZE) * BLOCK_SIZE;                     \
  for (int i = idx; i < rounded_size; i += BLOCK_SIZE * MAX_NUM_BLOCKS) {      \
    CURAND_T x = CURAND_FUNC(&state[blockIdx.x]);                                 \
    if (i < size) {                                                            \
      T y = TRANSFORM;                                                           \
      result[i] = y;                                                           \
    }                                                                          \
  }                                                                            \
}

#define GENERATE_KERNEL2(NAME, T, ARG1, ARG2, CURAND_T, CURAND_FUNC, TRANSFORM)                \
__global__ void NAME(hiprandStateMtgp32 *state, int size, T *result, ARG1, ARG2)  \
{                                                                                    \
  int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;                                   \
  int rounded_size = THCCeilDiv(size, BLOCK_SIZE) * BLOCK_SIZE;                           \
  for (int i = idx; i < rounded_size; i += BLOCK_SIZE * MAX_NUM_BLOCKS) {            \
    CURAND_T x = CURAND_FUNC(&state[blockIdx.x]);                                       \
    if (i < size) {                                                                  \
      T y = TRANSFORM;                                                                 \
      result[i] = y;                                                                 \
    }                                                                                \
  }                                                                                  \
}

GENERATE_KERNEL2(generate_uniform, float, double a, double b, float, hiprand_uniform, x * (b-a) + a)
GENERATE_KERNEL2(generate_uniform, double, double a, double b, double, hiprand_uniform_double, x * (b-a) + a)
GENERATE_KERNEL2(generate_uniform, half, double a, double b, float, hiprand_uniform, (ScalarConvert<float, half>::to(x * (b-a) + a)))

GENERATE_KERNEL1(generate_geometric, float, double p, float, hiprand_uniform, (log(1-x) / log(p)) + 1)

/* Separate kernel because hiprand_log_normal gets extra parameters. */
__global__ void generate_log_normal(hiprandStateMtgp32 *state, int size, float *result, float mean, float stddev)
{
  int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  int rounded_size = THCCeilDiv(size, BLOCK_SIZE) * BLOCK_SIZE;
  for (int i = idx; i < rounded_size; i += BLOCK_SIZE * MAX_NUM_BLOCKS) {
    float x = hiprand_log_normal(&state[blockIdx.x], mean, stddev);
    if (i < size) {
      result[i] = x;
    }
  }
}

#define NUM_BLOCKS min((int)THCCeilDiv(size, (ptrdiff_t) BLOCK_SIZE), MAX_NUM_BLOCKS)
THC_API void THCudaTensor_logNormal(THCState* state, THCudaTensor *self_, double mean, double stdv)
{
  THAssert(THCudaTensor_checkGPU(state, 1, self_));
  Generator* gen = THCRandom_getGenerator(state);

  THCudaTensor *self = THCudaTensor_newContiguous(state, self_);
  ptrdiff_t size = THCudaTensor_nElement(state, self);
  float *data = THCudaTensor_data(state, self);

  generate_log_normal<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->gen_states, size, data, mean, stdv);

  THCudaTensor_freeCopyTo(state, self, self_);
};

THC_API void THCudaTensor_geometric(THCState* state, THCudaTensor *self_, double p)
{
  THAssert(THCudaTensor_checkGPU(state, 1, self_));
  Generator* gen = THCRandom_getGenerator(state);

  THCudaTensor *self = THCudaTensor_newContiguous(state, self_);
  ptrdiff_t size = THCudaTensor_nElement(state, self);
  float *data = THCudaTensor_data(state, self);

  generate_geometric<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->gen_states, size, data, p);

  THCudaTensor_freeCopyTo(state, self, self_);
};

void THCudaTensor_renormRows(struct THCState* state,
                             THCudaTensor* t) {
  THAssert(THCudaTensor_nDimension(state, t) == 2);
  long rows = THCudaTensor_size(state, t, 0);
  long cols = THCudaTensor_size(state, t, 1);

  hipDeviceProp_t* props = THCState_getCurrentDeviceProperties(state);
  THAssert(props != NULL);

  int numSM = props->multiProcessorCount;
  int maxThreads = props->maxThreadsPerBlock;

  dim3 grid(rows < numSM * 4 ? rows : numSM * 4);
  dim3 block(cols < maxThreads ? cols : maxThreads);

  renormRowsL1<float>
    <<<grid, block, block.x * sizeof(float),
    THCState_getCurrentStream(state)>>>(THCudaTensor_data(state, t),
                                        rows, cols);
}

THC_API void THCudaTensor_multinomial(struct THCState *state,
                                      THCudaTensor *self,
                                      THCudaTensor *prob_dist,
                                      int n_sample,
                                      int with_replacement)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self, prob_dist));
  Generator* gen = THCRandom_getGenerator(state);

  int inputSize = THCudaTensor_nDimension(state, prob_dist);
  THArgCheck(inputSize > 0 && inputSize <= 2, 2,
             "prob_dist must be 1 or 2 dim");

  // Categories are in the innermost dimension
  long numDist =
    inputSize == 1 ? 1 : THCudaTensor_size(state, prob_dist, 0);
  long numCategoriesLong =
    inputSize == 1 ? THCudaTensor_size(state, prob_dist, 0) :
    THCudaTensor_size(state, prob_dist, 1);

  // Since the index tensor is float, numCategories cannot exceed max
  // float integer precision
  THArgCheck(numCategoriesLong <= FLOAT32_MAX_CONSECUTIVE_INT, 2,
             "number of categories cannot exceed 2^24");
  int numCategories = (int) numCategoriesLong;

  THArgCheck(n_sample > 0, 3, "cannot sample <= 0 samples");

  if (!with_replacement) {
    THArgCheck(n_sample <= numCategories, 2,
               "cannot sample n_sample > prob_dist:size(1) samples without "
               "replacement");
  }

  // It is possible that prob_dist is non-contiguous
  THCudaTensor* probDistContig =
    THCudaTensor_newContiguous(state, prob_dist);

  // Restructure data for 2d
  if (inputSize == 1) {
    THCudaTensor_resize2d(state, probDistContig, 1, numCategories);
  }

  THCudaTensor_resize2d(state, self, numDist, n_sample);

  if (n_sample == 1) {
    // Optimized allocation-free implementation

    // To exploit greater parallelism for the sampling, generate the
    // Uniform random samples in a separate kernel launch, into the
    // result memory. The device RNG is thread-limited
    THCudaTensor_uniform(state, self, 0.0, 1.0);

    hipDeviceProp_t* props = THCState_getCurrentDeviceProperties(state);
    THAssert(props != NULL);

    int numSM = props->multiProcessorCount;
    int maxThreads = props->maxThreadsPerBlock;

    dim3 block(numCategories < maxThreads ? numCategories : maxThreads);
    dim3 grid(numDist < numSM * 4 ? numDist : numSM * 4);

    sampleMultinomialOnce
      <<<grid, block, block.x * sizeof(float),
         THCState_getCurrentStream(state)>>>(
      THCudaTensor_data(state, self),
      numDist,
      numCategories,
      THCudaTensor_data(state, probDistContig));
  } else {
    // Generic, slow implementation with memory allocations

    // For sampling without replacement, we modify the distribution
    // for subsequent samples in this space
    THCudaTensor* origDist = THCudaTensor_new(state);
    THCudaTensor_resizeAs(state, origDist, probDistContig);
    THCudaTensor_copy(state, origDist, probDistContig);

    THCudaTensor* normDist = THCudaTensor_new(state);
    THCudaTensor_resizeAs(state, normDist, probDistContig);

    THCudaTensor* prefixSum = THCudaTensor_new(state);

    // Renorm along rows
    THCudaTensor_copy(state, normDist, origDist);
    THCudaTensor_renormRows(state, normDist);

    // Prefix sum along rows
    THCudaTensor_cumsum(state, prefixSum, normDist, 1);

    if (with_replacement) {
      // Sample with replacement

      // Binary search is warp divergent (so effectively we're running
      // with just a single thread), but for better utilization,
      // we need each block to have at least 4 warps.
      dim3 block(32, 4);

      // Each warp in a block will generate a sample from one
      // distribution concurrently.
      dim3 grid(numDist < MAX_NUM_BLOCKS ? numDist : MAX_NUM_BLOCKS);

      sampleMultinomialWithReplacement
        <<<grid, block, 0, THCState_getCurrentStream(state)>>>(
          gen->gen_states,
          n_sample,
          THCudaTensor_data(state, self),
          numDist, numCategories,
          THCudaTensor_data(state, prefixSum));
    } else {
      // Sample without replacement

      // Binary search is warp divergent (so effectively we're running
      // with just a single thread), but for better utilization,
      // we need each block to have at least 4 warps.
      dim3 block(32, 4);

      // Each warp in a block will generate a sample from a different
      // distribution concurrently.
      ptrdiff_t numBlocks = THCCeilDiv(numDist, 4L);
      dim3 grid(numBlocks < MAX_NUM_BLOCKS ? numBlocks : MAX_NUM_BLOCKS);

      for (int sample = 0; sample < n_sample; ++sample) {
        if (sample > 0) {
          // Update probabilities
          // Renorm along rows
          THCudaTensor_copy(state, normDist, origDist);
          THCudaTensor_renormRows(state, normDist);

          // Prefix sum along rows
          THCudaTensor_cumsum(state, prefixSum, normDist, 1);
        }

        // The kernel can only draw one sample before we have to
        // recalculate our distribution
        sampleMultinomialWithoutReplacement
          <<<grid, block, 0, THCState_getCurrentStream(state)>>>(
            gen->gen_states,
            n_sample,
            sample,
            THCudaTensor_data(state, self),
            numDist, numCategories,
            THCudaTensor_data(state, origDist),
            THCudaTensor_data(state, prefixSum));
      }
    }

    THCudaTensor_free(state, prefixSum);
    THCudaTensor_free(state, normDist);
    THCudaTensor_free(state, origDist);
  }

  // Revert data restructuring based on input sizes
  if (inputSize == 1) {
    THCudaTensor_resize1d(state, self, n_sample);

    // Unfortunately, if prob_dist is contiguous already,
    // newContiguous is not a private copy, so we have to restructure
    // this too, so as to not affect prob_dist
    THCudaTensor_resize1d(state, probDistContig, numCategories);
  }

  THCudaTensor_free(state, probDistContig);
}

#undef NUM_BLOCKS

#include "generic/THCTensorRandom.cu"
#include "THCGenerateAllTypes.h"

#undef GENERATE_KERNEL1
#undef GENERATE_KERNEL2

