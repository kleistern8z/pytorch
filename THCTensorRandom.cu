#include "hip/hip_runtime.h"
#include "THCTensorRandom.h"
#include "THCGeneral.h"

#include <thrust/functional.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_mtgp32_host.h>
#include <rocrand/rocrand_mtgp32_11213.h>

#define MAX_NUM_BLOCKS 64
#define BLOCK_SIZE 256

#ifndef DIVUP
#define DIVUP(x, y) (((x) + (y) - 1) / (y))
#endif

/* Sets up generator. Allocates but does not create the generator states. */
__host__ void initializeGenerator(Generator* gen)
{
  THCudaCheck(hipMalloc((void**)&gen->gen_states, MAX_NUM_BLOCKS * sizeof(hiprandStateMtgp32)));
  THCudaCheck(hipMalloc((void**)&gen->kernel_params, sizeof(mtgp32_kernel_params)));
  if (hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, gen->kernel_params) != HIPRAND_STATUS_SUCCESS)
  {
    THError("Creating MTGP constants failed.");
  }
}

/* Frees memory allocated during setup. */
__host__ void destroyGenerator(Generator* gen)
{
  if (gen->gen_states)
  {
    THCudaCheck(hipFree(gen->gen_states));
    gen->gen_states = NULL;
  }
  if (gen->kernel_params)
  {
    THCudaCheck(hipFree(gen->kernel_params));
    gen->kernel_params = NULL;
  }
}

/* Creates a new generator state given the seed. */
__host__ void createGeneratorState(Generator* gen, unsigned long seed)
{
  if (hiprandMakeMTGP32KernelState(gen->gen_states, mtgp32dc_params_fast_11213,
                                  gen->kernel_params, MAX_NUM_BLOCKS, seed) != HIPRAND_STATUS_SUCCESS)
  {
    THError("Creating MTGP kernel state failed.");
  }
}

/* Initialize generator array (must be called before any other function) */
__host__ void THCRandom_init(THCudaRNGState* state, int devices, int current_device)
{
  state->num_devices = devices;
  state->gen = (Generator*)malloc(state->num_devices * sizeof(Generator));
  for (int i = 0; i < state->num_devices; ++i)
  {
    state->gen[i].initf = 0;
    state->gen[i].initial_seed = 0;
    state->gen[i].gen_states = NULL;
    state->gen[i].kernel_params = NULL;
  }
  state->current_gen = &state->gen[current_device];
  // Initialize the generator for the current device. Other generators will be
  // initialized on-demand in THCRandom_setGenerator.
  initializeGenerator(state->current_gen);
  THCRandom_seed(state);
}

/* Destroy generators and free memory */
__host__ void THCRandom_shutdown(THCudaRNGState* state)
{
  if (state->gen == NULL) return;
  for (int i = 0; i < state->num_devices; ++i)
  {
    destroyGenerator(&state->gen[i]);
  }
  free(state->gen);
  state->gen = NULL;
  state->current_gen = NULL;
}

/* Set the generator for the current device */
__host__ void THCRandom_setGenerator(THCudaRNGState* state, int device)
{
  if (device >= state->num_devices) THError("Invalid device index.");
  state->current_gen = &state->gen[device];
  if (state->current_gen->initf == 0)
  {
    initializeGenerator(state->current_gen);
    THCRandom_seed(state);
  }
}

/* Reset the generator for the current device after a device reset */
__host__ void THCRandom_resetGenerator(THCudaRNGState* state)
{
  initializeGenerator(state->current_gen);
  THCRandom_manualSeed(state, state->current_gen->initial_seed);
}

/* Random seed */
__host__ unsigned long THCRandom_seed(THCudaRNGState* state)
{
  unsigned long s = (unsigned long)time(0);
  THCRandom_manualSeed(state, s);
  return s;
}

__host__ unsigned long THCRandom_seedAll(THCudaRNGState* state)
{
  unsigned long s = (unsigned long)time(0);
  THCRandom_manualSeedAll(state, s);
  return s;
}

/* Manually set the seed */
__host__ void THCRandom_manualSeed(THCudaRNGState* state, unsigned long seed)
{
  if (state->current_gen == NULL)
  {
    THError("Random number generators have not been initialized.");
  }
  state->current_gen->initial_seed = seed;
  createGeneratorState(state->current_gen, seed);
  state->current_gen->initf = 1;
}

__host__ void THCRandom_manualSeedAll(THCudaRNGState* state, unsigned long seed)
{
  int currentDevice;
  THCudaCheck(hipGetDevice(&currentDevice));
  for (int i = 0; i < state->num_devices; ++i) {
    THCudaCheck(hipSetDevice(i));
    THCRandom_setGenerator(state, i);
    THCRandom_manualSeed(state, seed);
  }
  THCudaCheck(hipSetDevice(currentDevice));
  THCRandom_setGenerator(state, currentDevice);
}

/* Get the initial seed */
__host__ unsigned long THCRandom_initialSeed(THCudaRNGState* state)
{
  return state->current_gen->initial_seed;
}

__host__ void THCRandom_getRNGState(THCudaRNGState* state, THByteTensor *rng_state)
{
  // The RNG state comprises the MTPG32 states and the seed.
  static const size_t states_size = MAX_NUM_BLOCKS * sizeof(hiprandStateMtgp32);
  static const size_t seed_size = sizeof(unsigned long);
  static const size_t total_size = states_size + seed_size;
  THByteTensor_resize1d(rng_state, total_size);
  THArgCheck(THByteTensor_nElement(rng_state) == total_size, 1, "RNG state is wrong size");
  THArgCheck(THByteTensor_isContiguous(rng_state), 1, "RNG state must be contiguous");
  THCudaCheck(hipMemcpy(THByteTensor_data(rng_state), state->current_gen->gen_states,
                         states_size, hipMemcpyDeviceToHost));
  memcpy(THByteTensor_data(rng_state) + states_size, &state->current_gen->initial_seed, seed_size);
}

__host__ void THCRandom_setRNGState(THCudaRNGState* state, THByteTensor *rng_state)
{
  static const size_t states_size = MAX_NUM_BLOCKS * sizeof(hiprandStateMtgp32);
  static const size_t seed_size = sizeof(unsigned long);
  static const size_t total_size = states_size + seed_size;
  THArgCheck(THByteTensor_nElement(rng_state) == total_size, 1, "RNG state is wrong size");
  THArgCheck(THByteTensor_isContiguous(rng_state), 1, "RNG state must be contiguous");
  THCudaCheck(hipMemcpy(state->current_gen->gen_states, THByteTensor_data(rng_state),
                         states_size, hipMemcpyHostToDevice));
  memcpy(&state->current_gen->initial_seed, THByteTensor_data(rng_state) + states_size, seed_size);
}

#define GENERATE_KERNEL1(NAME, ARG1, CURAND_FUNC, TRANSFORM)                   \
__global__ void NAME(hiprandStateMtgp32 *state, int size, float *result, ARG1)  \
{                                                                              \
  int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;                             \
  for (int i = idx; i < size; i += BLOCK_SIZE * MAX_NUM_BLOCKS) {              \
    float x = CURAND_FUNC(&state[blockIdx.x]);                                 \
    x = TRANSFORM;                                                             \
    result[i] = x;                                                             \
  }                                                                            \
}

#define GENERATE_KERNEL2(NAME, ARG1, ARG2, CURAND_FUNC, TRANSFORM)                   \
__global__ void NAME(hiprandStateMtgp32 *state, int size, float *result, ARG1, ARG2)  \
{                                                                                    \
  int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;                                   \
  for (int i = idx; i < size; i += BLOCK_SIZE * MAX_NUM_BLOCKS) {                    \
    float x = CURAND_FUNC(&state[blockIdx.x]);                                       \
    x = TRANSFORM;                                                                   \
    result[i] = x;                                                                   \
  }                                                                                  \
}

GENERATE_KERNEL2(generate_uniform, double a, double b, hiprand_uniform, x * (b-a) + a)
GENERATE_KERNEL1(generate_bernoulli, double p, hiprand_uniform, (float)x <= p)
GENERATE_KERNEL2(generate_normal, double mean, double stdv, hiprand_normal, (x * stdv) + mean)
GENERATE_KERNEL1(generate_geometric, double p, hiprand_uniform, (log(1-x) / log(p)) + 1)
GENERATE_KERNEL1(generate_exponential, double lambda, hiprand_uniform, (float)(-1. / lambda * log(1-x)))
GENERATE_KERNEL2(generate_cauchy, double median, double sigma, hiprand_uniform, (float)(median + sigma * tan(M_PI*(x-0.5))))

#undef GENERATE_KERNEL1
#undef GENERATE_KERNEL2

/* Separate kernel because hiprand_log_normal gets extra parameters. */
__global__ void generate_log_normal(hiprandStateMtgp32 *state, int size, float *result, float mean, float stddev)
{
  int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  for (int i = idx; i < size; i += BLOCK_SIZE * MAX_NUM_BLOCKS) {
    result[i] = hiprand_log_normal(&state[blockIdx.x], mean, stddev);
  }
}

#define NUM_BLOCKS min((int)DIVUP(size, BLOCK_SIZE), MAX_NUM_BLOCKS)
THC_API void THCudaTensor_uniform(THCudaRNGState* state, THCudaTensor *self_, double a, double b)
{
  if (state->current_gen == NULL)
  {
    THError("Random number generators have not been initialized.");
  }
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  float *data = THCudaTensor_data(self);

  generate_uniform<<<NUM_BLOCKS, BLOCK_SIZE>>>(
      state->current_gen->gen_states, size, data, a, b);

  THCudaTensor_freeCopyTo(self, self_);
};

THC_API void THCudaTensor_bernoulli(THCudaRNGState* state, THCudaTensor *self_, double p)
{
  if (state->current_gen == NULL)
  {
    THError("Random number generators have not been initialized.");
  }
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  float *data = THCudaTensor_data(self);

  generate_bernoulli<<<NUM_BLOCKS, BLOCK_SIZE>>>(
      state->current_gen->gen_states, size, data, p);

  THCudaTensor_freeCopyTo(self, self_);
};

THC_API void THCudaTensor_normal(THCudaRNGState* state, THCudaTensor *self_, double mean, double stdv)
{
  if (state->current_gen == NULL)
  {
    THError("Random number generators have not been initialized.");
  }
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  float *data = THCudaTensor_data(self);

  generate_normal<<<NUM_BLOCKS, BLOCK_SIZE>>>(
      state->current_gen->gen_states, size, data, mean, stdv);

  THCudaTensor_freeCopyTo(self, self_);
};

THC_API void THCudaTensor_logNormal(THCudaRNGState* state, THCudaTensor *self_, double mean, double stdv)
{
  if (state->current_gen == NULL)
  {
    THError("Random number generators have not been initialized.");
  }
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  float *data = THCudaTensor_data(self);

  generate_log_normal<<<NUM_BLOCKS, BLOCK_SIZE>>>(
      state->current_gen->gen_states, size, data, mean, stdv);

  THCudaTensor_freeCopyTo(self, self_);
};

THC_API void THCudaTensor_geometric(THCudaRNGState* state, THCudaTensor *self_, double p)
{
  if (state->current_gen == NULL)
  {
    THError("Random number generators have not been initialized.");
  }
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  float *data = THCudaTensor_data(self);

  generate_geometric<<<NUM_BLOCKS, BLOCK_SIZE>>>(
      state->current_gen->gen_states, size, data, p);

  THCudaTensor_freeCopyTo(self, self_);
};

THC_API void THCudaTensor_exponential(THCudaRNGState* state, THCudaTensor *self_, double lambda)
{
  if (state->current_gen == NULL)
  {
    THError("Random number generators have not been initialized.");
  }
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  float *data = THCudaTensor_data(self);

  generate_exponential<<<NUM_BLOCKS, BLOCK_SIZE>>>(
      state->current_gen->gen_states, size, data, lambda);

  THCudaTensor_freeCopyTo(self, self_);
};

THC_API void THCudaTensor_cauchy(THCudaRNGState* state, THCudaTensor *self_, double median, double sigma)
{
  if (state->current_gen == NULL)
  {
    THError("Random number generators have not been initialized.");
  }
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  float *data = THCudaTensor_data(self);

  generate_cauchy<<<NUM_BLOCKS, BLOCK_SIZE>>>(
      state->current_gen->gen_states, size, data, median, sigma);

  THCudaTensor_freeCopyTo(self, self_);
};
#undef NUM_BLOCKS
