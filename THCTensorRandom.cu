#include "hip/hip_runtime.h"
#include "THCTensorRandom.h"
#include "THCGeneral.h"

#include <thrust/random.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>

/* The initial seed. */
static unsigned long the_initial_seed = 0;
static int initf = 0;
__device__ static thrust::minstd_rand * rng = NULL;

/* Seeds */
__host__ unsigned long THCRandom_seed()
{
  unsigned long s = (unsigned long)1; // TODO: this should be random
  THCRandom_manualSeed(s);
  return s;
}

__host__ void THCRandom_manualSeed(unsigned long the_seed_)
{
  the_initial_seed = the_seed_;
  if (initf == 0) {
    hipMalloc(&rng, sizeof(thrust::minstd_rand));
    thrust::minstd_rand rnghost(the_initial_seed);
    hipMemcpy(rng, &rnghost, sizeof(thrust::minstd_rand), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    initf = 1;
  } else {
    rng->seed(the_initial_seed);
  }
}

__host__ unsigned long THCRandom_initialSeed()
{
  if(initf == 0) {
    THCRandom_seed();
  }
  return the_initial_seed;
}

__host__ __device__ unsigned long THCRandom_random()
{
  thrust::uniform_int_distribution<unsigned long> ufm(0,(((unsigned long)1)<<31)-1);
  return ufm(*rng);
}

/* generates a random number on [0,1)-double-interval */
__host__ __device__ static double __uniform__()
{
  thrust::uniform_real_distribution<double> ufm(0,1);
  return ufm(*rng);
}

__host__ __device__ unsigned long THCRandom_random1(long b)
{
  //THArgCheck(b > 0, 1, "upper bound must be strictly positive");
  return(THCRandom_random() % b + 1);
}

__host__ __device__ unsigned long THCRandom_random2(long a, long b)
{
  //THArgCheck(b >= a, 2, "upper bound must be larger than lower bound");
  return((THCRandom_random() % (b+1-a)) + a);
}

__host__ __device__ double THCRandom_uniform(double a, double b)
{
  return(__uniform__() * (b - a) + a);
}

__host__ __device__ double THCRandom_normal(double mean, double stdv)
{
  //THArgCheck(stdv > 0, 2, "standard deviation must be strictly positive");
  thrust::random::experimental::normal_distribution<double> normal(mean,stdv);
  return normal(*rng);
}

__host__ __device__ double THCRandom_exponential(double lambda)
{
  return(-1. / lambda * log(1-__uniform__()));
}

__host__ __device__ double THCRandom_cauchy(double median, double sigma)
{
  return(median + sigma * tan(M_PI*(__uniform__()-0.5)));
}

__host__ __device__ double THCRandom_logNormal(double mean, double stdv)
{
  double zm = mean*mean;
  double zs = stdv*stdv;
  //THArgCheck(stdv > 0, 2, "standard deviation must be strictly positive");
  return(exp(THCRandom_normal(log(zm/sqrt(zs + zm)), sqrt(log(zs/zm+1)) )));
}

__host__ __device__ int THCRandom_geometric(double p)
{
  //THArgCheck(p > 0 && p < 1, 1, "must be > 0 and < 1");
  return((int)(log(1-__uniform__()) / log(p)) + 1);
}

__host__ __device__ int THCRandom_bernoulli(double p)
{
  //THArgCheck(p > 0 && p < 1, 1, "must be > 0 and < 1");
  return(__uniform__() <= p);
}

struct random_functor
{
  random_functor() {}

  __host__ __device__ float operator()(const float& x) const
  {
    return (float)(THCRandom_random() % ((1UL << FLT_MANT_DIG)+1));
  }
};

TH_API void THCudaTensor_random(THCudaTensor *self_) { 
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));
  
  thrust::transform(self_data, self_data+size, self_data, random_functor());

  THCudaTensor_freeCopyTo(self, self_);
};

struct random1_functor
{
  const long b;

  random1_functor(long b_) : b(b_) {}

  __host__ __device__ float operator()(const float& x) const
  {
    return (float)(THCRandom_random() % b + 1);
  }
};

TH_API void THCudaTensor_random1(THCudaTensor *self_, long b) {
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));
  
  thrust::transform(self_data, self_data+size, self_data, random1_functor(b));

  THCudaTensor_freeCopyTo(self, self_);
};

struct random2_functor
{
  const long a,b;

  random2_functor(long a_, long b_) : a(a_), b(b_) {}

  __host__ __device__ float operator()(const float& x) const
  {
    return (float)((THCRandom_random() % (b+1-a)) + a);
  }
};

TH_API void THCudaTensor_random2(THCudaTensor *self_, long a, long b) {
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));
  
  thrust::transform(self_data, self_data+size, self_data, random2_functor(a,b));

  THCudaTensor_freeCopyTo(self, self_);
};

struct bernoulli_functor
{
  const double p;

  bernoulli_functor(double p_) : p(p_) {}

  __host__ __device__ float operator()(const float& x) const
  {
    return (float)(THCRandom_bernoulli(p));
  }
};

TH_API void THCudaTensor_bernoulli(THCudaTensor *self_, float p) {
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));
  
  thrust::transform(self_data, self_data+size, self_data, bernoulli_functor(p));

  THCudaTensor_freeCopyTo(self, self_);
};

struct uniform_functor
{
  const double a,b;

  uniform_functor(double a_, double b_) : a(a_),b(b_) {}

  __host__ __device__ float operator()(const float& x) const
  {
    return (float)(THCRandom_uniform(a,b));
  }
};

TH_API void THCudaTensor_uniform(THCudaTensor *self_, double a, double b) {
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));
  
  thrust::transform(self_data, self_data+size, self_data, uniform_functor(a,b));

  THCudaTensor_freeCopyTo(self, self_);
};

struct normal_functor
{
  const double mean,stdv;

  normal_functor(double mean_, double stdv_) : mean(mean_),stdv(stdv_) {}

  __host__ __device__ float operator()(const float& x) const
  {
    return (float)(THCRandom_normal(mean,stdv));
  }
};

TH_API void THCudaTensor_normal(THCudaTensor *self_, double mean, double stdv) {
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));
  
  thrust::transform(self_data, self_data+size, self_data, normal_functor(mean,stdv));

  THCudaTensor_freeCopyTo(self, self_);
};

// TODO: implement these guys:

TH_API void THCudaTensor_geometric(THCudaTensor *self_, float p) {
  //TH_TENSOR_APPLY(real, self, *self_data = (real)THCRandom_geometric(p););
};

TH_API void THCudaTensor_exponential(THCudaTensor *self_, double lambda) {
  //TH_TENSOR_APPLY(real, self, *self_data = (real)THCRandom_exponential(lambda););
};

TH_API void THCudaTensor_cauchy(THCudaTensor *self_, double median, double sigma) {
  //TH_TENSOR_APPLY(real, self, *self_data = (real)THCRandom_cauchy(median, sigma););
};

TH_API void THCudaTensor_logNormal(THCudaTensor *self_, double mean, double stdv) {
  //TH_TENSOR_APPLY(real, self, *self_data = (real)THCRandom_logNormal(mean, stdv););
};
