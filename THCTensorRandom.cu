#include "hip/hip_runtime.h"
#include "THCTensorRandom.h"
#include "THCGeneral.h"

#include <thrust/random.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>
#include <thrust/sequence.h>

/* The initial seed. */
__device__ static int initf = 0;
__device__ static unsigned long the_initial_seed = 0;
__device__ static unsigned long step = 0;

/* Seeds */
__host__ unsigned long THCRandom_seed()
{
  unsigned long s = (unsigned long)1; // TODO: this should be random
  THCRandom_manualSeed(s);
  return s;
}

__host__ void THCRandom_manualSeed(unsigned long the_seed_)
{
  the_initial_seed = the_seed_;
  initf = 1;
}

__host__ unsigned long THCRandom_initialSeed()
{
  if(initf == 0) THCRandom_seed();
  return the_initial_seed;
}

__host__ __device__ unsigned long THCRandom_random()
{
  thrust::default_random_engine rng(the_initial_seed); rng.discard(step++);
  thrust::uniform_int_distribution<unsigned long> ufm(0,(((unsigned long)1)<<31)-1);
  return ufm(rng);
}

/* generates a random number on [0,1)-double-interval */
__host__ __device__ static double __uniform__()
{
  thrust::default_random_engine rng(the_initial_seed); rng.discard(step++);
  thrust::uniform_real_distribution<double> ufm(0,1);
  return ufm(rng);
}

__host__ __device__ unsigned long THCRandom_random1(long b)
{
  //THArgCheck(b > 0, 1, "upper bound must be strictly positive");
  return(THCRandom_random() % b + 1);
}

__host__ __device__ unsigned long THCRandom_random2(long a, long b)
{
  //THArgCheck(b >= a, 2, "upper bound must be larger than lower bound");
  return((THCRandom_random() % (b+1-a)) + a);
}

__host__ __device__ double THCRandom_uniform(double a, double b)
{
  return(__uniform__() * (b - a) + a);
}

__host__ __device__ double THCRandom_normal(double mean, double stdv)
{
  //THArgCheck(stdv > 0, 2, "standard deviation must be strictly positive");
  thrust::default_random_engine rng(the_initial_seed); rng.discard(step++);
  thrust::random::experimental::normal_distribution<double> normal(mean,stdv);
  return normal(rng);
}

__host__ __device__ double THCRandom_exponential(double lambda)
{
  return(-1. / lambda * log(1-__uniform__()));
}

__host__ __device__ double THCRandom_cauchy(double median, double sigma)
{
  return(median + sigma * tan(M_PI*(__uniform__()-0.5)));
}

__host__ __device__ double THCRandom_logNormal(double mean, double stdv)
{
  //THArgCheck(stdv > 0, 2, "standard deviation must be strictly positive");
  double zm = mean*mean;
  double zs = stdv*stdv;
  thrust::default_random_engine rng(the_initial_seed); rng.discard(step++);
  thrust::random::experimental::normal_distribution<double> normal(log(zm/sqrt(zs + zm)), sqrt(log(zs/zm+1)));
  return exp(normal(rng));
}

__host__ __device__ int THCRandom_geometric(double p)
{
  //THArgCheck(p > 0 && p < 1, 1, "must be > 0 and < 1");
  return((int)(log(1-__uniform__()) / log(p)) + 1);
}

__host__ __device__ int THCRandom_bernoulli(double p)
{
  //THArgCheck(p > 0 && p < 1, 1, "must be > 0 and < 1");
  return(__uniform__() <= p);
}

struct random_functor
{
  random_functor() {}

  __host__ __device__ float operator()(const float& x) const
  {
    thrust::default_random_engine rng(the_initial_seed); rng.discard(x);
    thrust::uniform_int_distribution<unsigned long> ufm(0,(((unsigned long)1)<<31)-1);
    unsigned long r = ufm(rng);
    return (float)(r % ((1UL << FLT_MANT_DIG)+1));
  }
};

TH_API void THCudaTensor_random(THCudaTensor *self_) { 
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));

  thrust::sequence(self_data, self_data+size, step+=size);
  thrust::transform(self_data, self_data+size, self_data, random_functor());

  THCudaTensor_freeCopyTo(self, self_);
};

struct random1_functor
{
  const long b;

  random1_functor(long b_) : b(b_) {}

  __host__ __device__ float operator()(const float& x) const
  {
    thrust::default_random_engine rng(the_initial_seed); rng.discard(x);
    thrust::uniform_int_distribution<unsigned long> ufm(0,(((unsigned long)1)<<31)-1);
    unsigned long r = ufm(rng);
    return (float)(r % b + 1);
  }
};

TH_API void THCudaTensor_random1(THCudaTensor *self_, long b) {
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));

  thrust::sequence(self_data, self_data+size, step+=size);
  thrust::transform(self_data, self_data+size, self_data, random1_functor(b));

  THCudaTensor_freeCopyTo(self, self_);
};

struct random2_functor
{
  const long a,b;

  random2_functor(long a_, long b_) : a(a_), b(b_) {}

  __host__ __device__ float operator()(const float& x) const
  {
    thrust::default_random_engine rng(the_initial_seed); rng.discard(x);
    thrust::uniform_int_distribution<unsigned long> ufm(0,(((unsigned long)1)<<31)-1);
    unsigned long r = ufm(rng);
    return (float)((r % (b+1-a)) + a);
  }
};

TH_API void THCudaTensor_random2(THCudaTensor *self_, long a, long b) {
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));
  
  thrust::sequence(self_data, self_data+size, step+=size);
  thrust::transform(self_data, self_data+size, self_data, random2_functor(a,b));

  THCudaTensor_freeCopyTo(self, self_);
};

struct bernoulli_functor
{
  const double p;

  bernoulli_functor(double p_) : p(p_) {}

  __host__ __device__ float operator()(const float& x) const
  {
    thrust::default_random_engine rng(the_initial_seed); rng.discard(x);
    thrust::uniform_real_distribution<float> uniform(0,1);
    return (float)(uniform(rng) <= p);
  }
};

TH_API void THCudaTensor_bernoulli(THCudaTensor *self_, double p) {
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));
  
  thrust::sequence(self_data, self_data+size, step+=size);
  thrust::transform(self_data, self_data+size, self_data, bernoulli_functor(p));

  THCudaTensor_freeCopyTo(self, self_);
};

struct uniform_functor
{
  const double a,b;

  uniform_functor(double a_, double b_) : a(a_),b(b_) {}

  __host__ __device__ float operator()(const float& x) const
  {
    thrust::default_random_engine rng(the_initial_seed); rng.discard(x);
    thrust::uniform_real_distribution<float> uniform(a,b);
    return uniform(rng);
  }
};

TH_API void THCudaTensor_uniform(THCudaTensor *self_, double a, double b) {
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));
  
  thrust::sequence(self_data, self_data+size, step+=size);
  thrust::transform(self_data, self_data+size, self_data, uniform_functor(a,b));

  THCudaTensor_freeCopyTo(self, self_);
};

struct normal_functor
{
  const double mean,stdv;

  normal_functor(double mean_, double stdv_)
    : mean(mean_),stdv(stdv_) {}

  __host__ __device__ 
  float operator()(const float& x) const
  {
    thrust::default_random_engine rng(the_initial_seed); rng.discard(x);
    thrust::random::experimental::normal_distribution<float> normal(mean,stdv);
    return normal(rng);
  }
};

TH_API void THCudaTensor_normal(THCudaTensor *self_, double mean, double stdv) {
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));

  thrust::sequence(self_data, self_data+size, step+=size);
  thrust::transform(self_data, self_data+size, self_data, normal_functor(mean,stdv));

  THCudaTensor_freeCopyTo(self, self_);
};

struct geometric_functor
{
  const double p;

  geometric_functor(double p_) : p(p_) {}

  __host__ __device__ float operator()(const float& x) const
  {
    thrust::default_random_engine rng(the_initial_seed); rng.discard(x);
    thrust::uniform_real_distribution<float> uniform(0,1);
    float u = uniform(rng);
    return (float)((log(1-u) / log(p)) + 1);
  }
};

TH_API void THCudaTensor_geometric(THCudaTensor *self_, double p) {
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));
  
  thrust::sequence(self_data, self_data+size, step+=size);
  thrust::transform(self_data, self_data+size, self_data, geometric_functor(p));

  THCudaTensor_freeCopyTo(self, self_);
};

struct exponential_functor
{
  const double lambda;

  exponential_functor(double lambda_) : lambda(lambda_) {}

  __host__ __device__ float operator()(const float& x) const
  {
    thrust::default_random_engine rng(the_initial_seed); rng.discard(x);
    thrust::uniform_real_distribution<float> uniform(0,1);
    float u = uniform(rng);
    return (float)(-1. / lambda * log(1-u));
  }
};

TH_API void THCudaTensor_exponential(THCudaTensor *self_, double lambda) {
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));
  
  thrust::sequence(self_data, self_data+size, step+=size);
  thrust::transform(self_data, self_data+size, self_data, exponential_functor(lambda));

  THCudaTensor_freeCopyTo(self, self_);
};

struct cauchy_functor
{
  const double median,sigma;

  cauchy_functor(double median_, double sigma_) : median(median_),sigma(sigma_) {}

  __host__ __device__ float operator()(const float& x) const
  {
    thrust::default_random_engine rng(the_initial_seed); rng.discard(x);
    thrust::uniform_real_distribution<float> uniform(0,1);
    float u = uniform(rng);
    return (float)(median + sigma * tan(M_PI*(u-0.5)));
  }
};

TH_API void THCudaTensor_cauchy(THCudaTensor *self_, double median, double sigma) {
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));
  
  thrust::sequence(self_data, self_data+size, step+=size);
  thrust::transform(self_data, self_data+size, self_data, cauchy_functor(median, sigma));

  THCudaTensor_freeCopyTo(self, self_);
};

struct logNormal_functor
{
  const double mean,stdv;

  logNormal_functor(double mean_, double stdv_) : mean(mean_),stdv(stdv_) {}

  __host__ __device__ float operator()(const float& x) const
  {
    double zm = mean*mean;
    double zs = stdv*stdv;
    thrust::default_random_engine rng(the_initial_seed); rng.discard(x);
    thrust::random::experimental::normal_distribution<double> normal(log(zm/sqrt(zs + zm)), sqrt(log(zs/zm+1)));
    return exp(normal(rng));
  }
};

TH_API void THCudaTensor_logNormal(THCudaTensor *self_, double mean, double stdv) {
  THCudaTensor *self = THCudaTensor_newContiguous(self_);
  long size = THCudaTensor_nElement(self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(self));
  
  thrust::sequence(self_data, self_data+size, step+=size);
  thrust::transform(self_data, self_data+size, self_data, logNormal_functor(mean, stdv));

  THCudaTensor_freeCopyTo(self, self_);
};
