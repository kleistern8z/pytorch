#include "hip/hip_runtime.h"
#include "utils.h"
#include "common.h"

// kernels borrowed from Caffe
template <typename Dtype>
__global__ void MaxPoolForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w, Dtype* top_data,
    Dtype* top_mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + kernel_h, height);
    int wend = min(wstart + kernel_w, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    Dtype maxval = -FLT_MAX;
    int maxidx = -1;
    bottom_data += (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        if (bottom_data[h * width + w] > maxval) {
          maxidx = h * width + w;
          maxval = bottom_data[maxidx];
        }
      }
    }
    top_data[index] = maxval;
    top_mask[index] = maxidx + 1;
  }
}


template <typename Dtype>
__global__ void MaxPoolBackward(const int nthreads, const Dtype* top_diff,
    const Dtype* top_mask, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int phstart =
        (h + pad_h < kernel_h) ? 0 : (h + pad_h - kernel_h) / stride_h + 1;
    int phend = min((h + pad_h) / stride_h + 1, pooled_height);
    int pwstart =
        (w + pad_w < kernel_w) ? 0 : (w + pad_w - kernel_w) / stride_w + 1;
    int pwend = min((w + pad_w) / stride_w + 1, pooled_width);
    Dtype gradient = 0;
    int offset = (n * channels + c) * pooled_height * pooled_width;
    top_diff += offset;
    top_mask += offset;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
	if (top_mask[ph * pooled_width + pw] - 1 == h * width + w) {
	  gradient += top_diff[ph * pooled_width + pw];
	}
      }
    }
    bottom_diff[index] = gradient;
  }
}

static int cunn_SpatialMaxPooling_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  int padW = luaT_getfieldcheckint(L, 1, "padW");
  int padH = luaT_getfieldcheckint(L, 1, "padH");
  bool ceil_mode = luaT_getfieldcheckboolean(L, 1, "ceil_mode");

  THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THCudaTensor *indices = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "indices", "torch.CudaTensor");

  THAssert(THCudaTensor_checkGPU(state, 3, input, output, indices));
  luaL_argcheck(L, input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch) tensor expected");

  long nInputCols, nInputRows, nInputPlane, batchSize;
  long nOutputCols, nOutputRows;

  if (input->nDimension == 3) {
    nInputCols = input->size[2];
    nInputRows = input->size[1];
    nInputPlane = input->size[0];
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size[3];
    nInputRows = input->size[2];
    nInputPlane = input->size[1];
    batchSize = input->size[0];
  }

  luaL_argcheck(L, nInputCols >= kW - padW && nInputRows >= kH - padH, 2, "input image smaller than kernel size");
  luaL_argcheck(L, kW/2 >= padW && kH/2 >= padH, 2, "pad should be smaller than half of kernel size");

  if(ceil_mode) {
    nOutputCols = ceil(float(nInputCols - kW + 2*padW) / float(dW)) + 1;
    nOutputRows = ceil(float(nInputRows - kH + 2*padH) / float(dH)) + 1;
  }
  else {
    nOutputCols = floor(float(nInputCols - kW + 2*padW) / float(dW)) + 1;
    nOutputRows = floor(float(nInputRows - kH + 2*padH) / float(dH)) + 1;
  }  
  
  if (padW || padH)
  {
    // ensure that the last pooling starts inside the image
    if ((nOutputRows - 1)*dH >= nInputRows + padH)
      --nOutputRows;
    if ((nOutputCols  - 1)*dW >= nInputCols  + padW)
      --nOutputCols;
  }

  input = THCudaTensor_newContiguous(state, input);
  float* input_data = THCudaTensor_data(state, input);

  THCudaTensor_resize4d(state, output, batchSize, nInputPlane, nOutputRows, nOutputCols);
  THCudaTensor_resizeAs(state, indices, output);
  
  float* indices_data = THCudaTensor_data(state, indices);
  float* output_data = THCudaTensor_data(state, output);

  int count = THCudaTensor_nElement(state, output);

  MaxPoolForward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count, input_data,
      batchSize, nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
      kH, kW, dH, dW, padH, padW, output_data, indices_data);

  if(input->nDimension == 3)
    THCudaTensor_resize3d(state, output, nInputPlane, nOutputRows, nOutputCols);

  THCudaTensor_free(state, input);

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in SpatialMaxPooling.updateOutput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  return 1;
}

static int cunn_SpatialMaxPooling_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  int padW = luaT_getfieldcheckint(L, 1, "padW");
  int padH = luaT_getfieldcheckint(L, 1, "padH");
  bool ceil_mode = luaT_getfieldcheckboolean(L, 1, "ceil_mode");

  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  THCudaTensor *indices = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "indices", "torch.CudaTensor");

  THAssert(THCudaTensor_checkGPU(state, 4, input, gradOutput, indices, gradInput));

  input = THCudaTensor_newContiguous(state, input);
  gradOutput = THCudaTensor_newContiguous(state, gradOutput);

  long nInputCols, nInputRows, nInputPlane, batchSize;
  long nOutputCols, nOutputRows;

  if (input->nDimension == 3) {
    nInputCols = input->size[2];
    nInputRows = input->size[1];
    nInputPlane = input->size[0];
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size[3];
    nInputRows = input->size[2];
    nInputPlane = input->size[1];
    batchSize = input->size[0];
  }

  if(ceil_mode) {
    nOutputCols = ceil(float(nInputCols - kW + 2*padW) / float(dW)) + 1;
    nOutputRows = ceil(float(nInputRows - kH + 2*padH) / float(dH)) + 1;
  }
  else {
    nOutputCols = floor(float(nInputCols - kW + 2*padW) / float(dW)) + 1;
    nOutputRows = floor(float(nInputRows - kH + 2*padH) / float(dH)) + 1;
  }


  gradOutput = THCudaTensor_newContiguous(state, gradOutput);
  THCudaTensor_resizeAs(state, gradInput, input);
  
  int count = THCudaTensor_nElement(state, input);

  MaxPoolBackward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>> 
      (count,
      THCudaTensor_data(state, gradOutput),
      THCudaTensor_data(state, indices),
      batchSize, nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
      kH, kW, dH, dW, padH, padW,
      THCudaTensor_data(state, gradInput));

  THCudaTensor_free(state, gradOutput);

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in SpatialMaxPooling.updateGradInput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  // clean
  THCudaTensor_free(state, input);
  THCudaTensor_free(state, gradOutput);

  return 1;
}

static const struct luaL_Reg cunn_SpatialMaxPooling__ [] = {
  {"SpatialMaxPooling_updateOutput", cunn_SpatialMaxPooling_updateOutput},
  {"SpatialMaxPooling_updateGradInput", cunn_SpatialMaxPooling_updateGradInput},
  {NULL, NULL}
};

void cunn_SpatialMaxPooling_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_SpatialMaxPooling__, "nn");
  lua_pop(L,1);
}
