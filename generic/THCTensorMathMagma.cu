
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorMathMagma.cu"
#else

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)

THC_API void THCTensor_(gesv)(THCState *state, THCTensor *rb_, THCTensor *ra_, THCTensor *b_, THCTensor *a_)
{
#ifdef USE_MAGMA
  THArgCheck(a_->nDimension == 2, 1, "A should be 2 dimensional");
  THArgCheck(b_->nDimension == 2, 2, "b should be 2 dimensional");
  THArgCheck(a_->size[0] == a_->size[1], 1, "A should be square");
  THArgCheck(b_->size[0] == a_->size[0], 2, "A,b size incompatible");

  int n = a_->size[0];
  int nrhs = b_->size[1];

  THCTensor *a = THCTensor_(newColumnMajor)(state, ra_, a_);
  THCTensor *b = THCTensor_(newColumnMajor)(state, rb_, b_);
  real *a_data = THCTensor_(data)(state, a);
  real *b_data = THCTensor_(data)(state, b);

  int *ipiv = th_magma_malloc_pinned<int>(n);

  int info;
#if defined(THC_REAL_IS_FLOAT)
  magma_sgesv_gpu(n, nrhs, a_data, n, ipiv, b_data, n, &info);
#else
  magma_dgesv_gpu(n, nrhs, a_data, n, ipiv, b_data, n, &info);
#endif

  if (info < 0)
    THError("MAGMA gesv : Argument %d : illegal value", -info);
  else if (info > 0)
    THError("MAGMA gesv : U(%d,%d) is zero, singular U.", info, info);

  magma_free_pinned(ipiv);
  THCTensor_(freeCopyTo)(state, a, ra_);
  THCTensor_(freeCopyTo)(state, b, rb_);
#else
  THError(NoMagma(gesv));
#endif
}

void THCTensor_(gels)(THCState *state, THCTensor *rb_, THCTensor *ra_, THCTensor *b_, THCTensor *a_)
{
#ifdef USE_MAGMA
  THArgCheck(a_->nDimension == 2, 1, "A should be 2 dimensional");
  THArgCheck(b_->nDimension == 2, 1, "b should be 2 dimensional");
  THArgCheck(a_->size[0] == b_->size[0], 2, "size incompatible A,b");
  THArgCheck(a_->size[0] >= a_->size[1], 2, "A should have m >= n");

  THCTensor *a = THCTensor_(newColumnMajor)(state, ra_, a_);
  THCTensor *b = THCTensor_(newColumnMajor)(state, rb_, b_);
  real *a_data = THCTensor_(data)(state, a);
  real *b_data = THCTensor_(data)(state, b);

  int m = a->size[0];
  int n = a->size[1];
  int nrhs = b->size[1];
  real wkopt;

  int info;
#if defined(THC_REAL_IS_FLOAT)
  magma_sgels_gpu(MagmaNoTrans, m, n, nrhs, a_data, m, b_data, m, &wkopt, -1, &info);
#else
  magma_dgels_gpu(MagmaNoTrans, m, n, nrhs, a_data, m, b_data, m, &wkopt, -1, &info);
#endif

  real *hwork = th_magma_malloc_pinned<real>((size_t)wkopt);

#if defined(THC_REAL_IS_FLOAT)
  magma_sgels_gpu(MagmaNoTrans, m, n, nrhs, a_data, m, b_data, m, hwork, (int)wkopt, &info);
#else
  magma_dgels_gpu(MagmaNoTrans, m, n, nrhs, a_data, m, b_data, m, hwork, (int)wkopt, &info);
#endif

  magma_free_pinned(hwork);

  if (info != 0)
    THError("MAGMA gels : Argument %d : illegal value", -info);

  THCTensor_(freeCopyTo)(state, a, ra_);
  THCTensor_(freeCopyTo)(state, b, rb_);
#else
  THError(NoMagma(gels));
#endif
}

#endif

#endif
