
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorMathMagma.cu"
#else

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)

THC_API void THCTensor_(gesv)(THCState *state, THCTensor *rb_, THCTensor *ra_, THCTensor *b_, THCTensor *a_)
{
#ifdef USE_MAGMA
  THArgCheck(a_->nDimension == 2, 1, "A should be 2 dimensional");
  THArgCheck(b_->nDimension == 2, 2, "b should be 2 dimensional");
  THArgCheck(a_->size[0] == a_->size[1], 1, "A should be square");
  THArgCheck(b_->size[0] == a_->size[0], 2, "A,b size incompatible");

  int n = a_->size[0];
  int nrhs = b_->size[1];

  THCTensor *a = THCTensor_(newColumnMajor)(state, ra_, a_);
  THCTensor *b = THCTensor_(newColumnMajor)(state, rb_, b_);
  real *a_data = THCTensor_(data)(state, a);
  real *b_data = THCTensor_(data)(state, b);

  int *ipiv = th_magma_malloc_pinned<int>(n);

  int info;
#if defined(THC_REAL_IS_FLOAT)
  magma_sgesv_gpu(n, nrhs, a_data, n, ipiv, b_data, n, &info);
#else
  magma_dgesv_gpu(n, nrhs, a_data, n, ipiv, b_data, n, &info);
#endif

  if (info < 0)
    THError("MAGMA gesv : Argument %d : illegal value", -info);
  else if (info > 0)
    THError("MAGMA gesv : U(%d,%d) is zero, singular U.", info, info);

  magma_free_pinned(ipiv);
  THCTensor_(freeCopyTo)(state, a, ra_);
  THCTensor_(freeCopyTo)(state, b, rb_);
#else
  THError(NoMagma(gesv));
#endif
}

#endif

#endif
