
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorMath.cu"
#else

THC_API void
THCTensor_(fill)(THCState* state, THCTensor *self_, real value)
{
  THAssert(THCTensor_(checkGPU)(state, 1, self_));

  if (!THC_pointwiseApply1(
        state, self_, TensorFillOp<real>(value))) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(zero)(THCState *state, THCTensor *self_)
{
  THAssert(THCTensor_(checkGPU)(state, 1, self_));
  if (THCTensor_(isContiguous)(state, self_)) {
    THCudaCheck(cudaMemsetAsync(THCTensor_(data)(state, self_),
                                0,
                                sizeof(real) * THCTensor_(nElement)(state, self_),
                                THCState_getCurrentStream(state)));
  } else {
    if (!THC_pointwiseApply1(
          state, self_,
          TensorFillOp<real>(ScalarConvert<int, real>::to(0)))) {
      THArgCheck(false, 1, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(zeros)(THCState *state, THCTensor *r_, THLongStorage *size)
{
  THAssert(THCTensor_(checkGPU)(state, 1, r_));
  THCTensor_(resize)(state, r_, size, NULL);
  THCTensor_(zero)(state, r_);
}

THC_API void
THCTensor_(ones)(THCState *state, THCTensor *r_, THLongStorage *size)
{
  THAssert(THCTensor_(checkGPU)(state, 1, r_));
  THCTensor_(resize)(state, r_, size, NULL);
  THCTensor_(fill)(state, r_, ScalarConvert<int, real>::to(1));
}

THC_API void
THCTensor_(reshape)(THCState *state, THCTensor *r_, THCTensor *t, THLongStorage *size)
{
  THAssert(THCTensor_(checkGPU)(state, 2, r_, t));
  THCTensor_(resize)(state, r_, size, NULL);
  THCTensor_(copy)(state, r_, t);
}

ptrdiff_t
THCTensor_(numel)(THCState *state, THCTensor *t)
{
  return THCTensor_(nElement)(state, t);
}

void THCTensor_(cat)(THCState *state, THCTensor *result,
		     THCTensor *ta, THCTensor *tb, int dimension)
{
  THCTensor* inputs[2];
  inputs[0] = ta;
  inputs[1] = tb;
  THCTensor_(catArray)(state, result, inputs, 2, dimension);
}

void THCTensor_(catArray)(THCState *state, THCTensor *result,
			  THCTensor **inputs, int numInputs, int dimension)
{
  THLongStorage *size;
  int i, j;
  long offset;
  int ndim = dimension + 1;
  for (i = 0; i < numInputs; i++)
  {
    ndim = THMax(ndim, THCTensor_(nDimension)(state, inputs[i]));
  }

  THArgCheck(numInputs > 0, 3, "invalid number of inputs %d", numInputs);
  THArgCheck(dimension >= 0, 4, "invalid dimension %d", dimension+1);

  size = THLongStorage_newWithSize(ndim);
  for(i = 0; i < ndim; i++)
  {
    long dimSize = i < THCTensor_(nDimension)(state, inputs[0])
                       ? THCTensor_(size)(state, inputs[0], i)
                       : 1;
    if (i == dimension)
    {
      for (j = 1; j < numInputs; j++)
      {
        dimSize += i < THCTensor_(nDimension)(state, inputs[j])
                       ? THCTensor_(size)(state, inputs[j], i)
                       : 1;
      }
    }
    else
    {
      for (j = 1; j < numInputs; j++)
      {
        if (dimSize != (i < THCTensor_(nDimension)(state, inputs[j])
			? THCTensor_(size)(state, inputs[j], i)
			: 1)) {
          THLongStorage_free(size);
          THError("inconsistent tensor sizes");
        }
      }
    }
    size->data[i] = dimSize;
  }

  THCTensor_(resize)(state, result, size, NULL);
  THLongStorage_free(size);

  offset = 0;
  for (j = 0; j < numInputs; j++)
  {
    long dimSize = dimension < THCTensor_(nDimension)(state, inputs[j])
			       ? THCTensor_(size)(state, inputs[j], dimension)
			       : 1;
    THCTensor *nt = THCTensor_(newWithTensor)(state, result);
    THCTensor_(narrow)(state, nt, NULL, dimension, offset, dimSize);
    THCTensor_(copy)(state, nt, inputs[j]);
    THCTensor_(free)(state, nt);
    offset += dimSize;
  }
}

void THCTensor_(nonzero)(THCState* state, THCudaLongTensor *tensor,
                          THCTensor *self)
{
  THAssert(THCTensor_(checkGPU)(state, 1, self  ));
  THAssert(THCudaLongTensor_checkGPU(state, 1, tensor));

  using namespace thrust::placeholders;

  self = THCTensor_(newContiguous)(state, self);
  thrust::device_ptr<real> self_data(THCTensor_(data)(state, self));

  int num_dim = THCTensor_(nDimension)(state, self);
  long N = THCTensor_(nElement)(state, self);

  THCudaLongTensor_resize2d(state, tensor, N, num_dim);
  tensor = THCudaLongTensor_newContiguous(state, tensor);
  thrust::device_ptr<long> tensor_data(THCudaLongTensor_data(state, tensor));

  thrust::counting_iterator<long> idxfirst(0);
  thrust::counting_iterator<long> idxlast = idxfirst + N;

  typedef thrust::device_ptr<long> Iter;
  strided_range<Iter> strided_tensor(tensor_data,
                                     tensor_data+N*num_dim, num_dim);

#if CUDA_VERSION >= 7000
  cudaStream_t stream = THCState_getCurrentStream(state);
#endif

  strided_range<Iter>::iterator dend = thrust::copy_if(
#if CUDA_VERSION >= 7000
    thrust::cuda::par.on(stream),
#endif
    idxfirst,
    idxlast,
    self_data,
    strided_tensor.begin(),
    NonZeroOp<real>()
  );

  long num_nonzeros = thrust::distance(strided_tensor.begin(), dend);

  long div = 1;
  for (int dim = num_dim-1; dim >= 0; dim--) {
    strided_range<Iter> stride_dim(tensor_data+dim,
                                   tensor_data+N*num_dim, num_dim);
    thrust::transform(
#if CUDA_VERSION >= 7000
      thrust::cuda::par.on(stream),
#endif
      strided_tensor.begin(),
      strided_tensor.end(),
      stride_dim.begin(),
      idx_functor(div, self->size[dim])
    );
    div *= self->size[dim];
  }

  THCudaLongTensor_resize2d(state, tensor, num_nonzeros, num_dim);

  THCTensor_(free)(state, self);
  THCudaLongTensor_free(state, tensor);

  THCudaCheck(cudaGetLastError());
}

#endif
