#include "hip/hip_runtime.h"
#include "utils.h"
#include "common.h"


#define CUDA_KERNEL_LOOP(i, n) \
for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
  i < (n); \
  i += blockDim.x * gridDim.x)

#define CUDA_CHECK(L, condition) \
/* Code block avoids redefinition of hipError_t error */ \
 do { \
   hipError_t error = condition; \
   luaL_argcheck(L, error == hipSuccess, 2, hipGetErrorString(error)); \
 } while (0)

template <typename Dtype>
__global__ void vol2col_kernel(const int n, const Dtype* data_im,
    const int length, const int height, const int width, const int ksize, const int kdepth, const int pad,
    const int temporal_pad, const int stride, const int temporal_stride, const int length_col, const int height_col, const int width_col,
    Dtype* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    int w_out = index % width_col;
    int h_out = (index / width_col ) % height_col;
    int l_out = (index / width_col / height_col) % length_col;
    int channel_in = index / width_col / height_col / length_col;
    int channel_out = channel_in * kdepth * ksize * ksize;
    int h_in = h_out * stride - pad;
    int w_in = w_out * stride - pad;
    int l_in = l_out * temporal_stride - temporal_pad;

    data_col += ((channel_out * length_col + l_out) * height_col + h_out) * width_col + w_out;
    data_im += ((channel_in * length + l_in) * height + h_in) * width + w_in;
    for (int k = 0; k < kdepth; ++k) {
      for (int i = 0; i < ksize; ++i) {
        for (int j = 0; j < ksize; ++j) {
          int l = l_in + k;
          int h = h_in + i;
          int w = w_in + j;
          *data_col = (l >= 0 && h >= 0 && w >= 0 && h < height && w < width && l < length) ?
              data_im[(k * height + i) * width + j] : 0;
          data_col += length_col * height_col * width_col;
        }
      }
    }
  }
}

template <typename Dtype>
void vol2col(const Dtype* data_im, const int channels, const int length,
    const int height, const int width, const int ksize, const int kdepth, const int pad,
    const int temporal_pad, const int stride, const int temporal_stride, Dtype* data_col) {

  int length_col = (length + 2 * temporal_pad - kdepth) / temporal_stride + 1;
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int num_kernels = channels * length_col * height_col * width_col;

  vol2col_kernel<Dtype><<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS>>>(
      num_kernels, data_im, length, height, width, ksize, kdepth, pad, temporal_pad, stride, temporal_stride,
      length_col, height_col, width_col, data_col);

}

// Explicit instantiation
template void vol2col<float>(const float* data_im, const int channels, const int length,
    const int height, const int width, const int ksize, const int kdepth, const int pad,
    const int temporal_pad, const int stride, const int temporal_stride, float* data_col);
template void vol2col<double>(const double* data_im, const int channels, const int length,
    const int height, const int width, const int ksize, const int kdepth, const int pad,
    const int temporal_pad, const int stride, const int temporal_stride, double* data_col);

template <typename Dtype>
__global__ void col2vol_kernel(const int n, const Dtype* data_col,
    const int length, const int height, const int width, const int channels, const int ksize, const int kdepth,
    const int pad, const int temporal_pad, const int stride, const int temporal_stride, const int length_col, const int height_col, const int width_col,
    Dtype* data_im) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = 0;
    int w = index % width + pad;
    int h = (index / width) % height + pad;
    int l = (index / width / height) % length + temporal_pad;
    int c = index / (width * height * length);
    // compute the start and end of the output
    int w_col_start = (w < ksize) ? 0 : (w - ksize) / stride + 1;
    int w_col_end = min(w / stride + 1, width_col);
    int h_col_start = (h < ksize) ? 0 : (h - ksize) / stride + 1;
    int h_col_end = min(h / stride + 1, height_col);
    int l_col_start = (l < kdepth) ? 0 : (l - kdepth) / temporal_stride + 1;
    int l_col_end = min(l / temporal_stride + 1, length_col);

    int offset = (c * kdepth * ksize * ksize + l * ksize * ksize + h * ksize + w) * length_col * height_col * width_col;

    int coeff_l_col = (1 - temporal_stride * ksize * ksize * length_col) * height_col * width_col;
    int coeff_h_col = (1 - stride * ksize * length_col * height_col) * width_col;
    int coeff_w_col = (1 - stride * length_col * height_col * width_col);

    for (int l_col = l_col_start; l_col < l_col_end; ++l_col) {
      for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
        for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
          val += data_col[offset + l_col * coeff_l_col + h_col * coeff_h_col + w_col * coeff_w_col];
        }
      }
    }
    data_im[index] = val;
  }
}

template <typename Dtype>
void col2vol(const Dtype* data_col, const int channels, const int length,
    const int height, const int width, const int ksize, const int kdepth, const int pad,
    const int temporal_pad, const int stride, const int temporal_stride, Dtype* data_im) {

  int length_col = (length + 2 * temporal_pad - kdepth) / temporal_stride + 1;
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int num_kernels = channels * length * height * width;

  col2vol_kernel<Dtype><<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS>>>(
      num_kernels, data_col, length, height, width, channels, ksize, kdepth, pad, temporal_pad, stride, temporal_stride,
      length_col, height_col, width_col, data_im);
}

// Explicit instantiation
template void col2vol<float>(const float* data_col, const int channels, const int length,
    const int height, const int width, const int ksize, const int kdepth, const int pad,
    const int temporal_pad, const int stride, const int temporal_stride, float* data_im);
template void col2vol<double>(const double* data_col, const int channels, const int length,
    const int height, const int width, const int ksize, const int kdepth, const int pad,
    const int temporal_pad, const int stride, const int temporal_stride, double* data_im);

static int cunn_VolumetricFullConvolution_updateOutput(lua_State *L) {
  THCState *state = getCutorchState(L);

  // Input
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  // Params:
  int dT = luaT_getfieldcheckint(L, 1, "dT");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int kT = luaT_getfieldcheckint(L, 1, "kT");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int pT = luaT_getfieldcheckint(L, 1, "pT");
  int pH = luaT_getfieldcheckint(L, 1, "pH");
  int pW = luaT_getfieldcheckint(L, 1, "pW");
  int nInputPlane = luaT_getfieldcheckint(L, 1, "nInputPlane");
  int nOutputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");

  THCudaTensor *weight = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *bias = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "bias", "torch.CudaTensor");
  THCudaTensor *columns = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "finput", "torch.CudaTensor");
  THCudaTensor *ones = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "fgradInput", "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");

  int inputDepth   = input->size[2];
  int inputHeight  = input->size[3];
  int inputWidth   = input->size[4];

  int outputDepth  = (inputDepth - 1) * dT - 2 * pT + kT;
  int outputHeight = (inputHeight - 1) * dH - 2 * pH + kH;
  int outputWidth  = (inputWidth - 1) * dW - 2 * pW + kW;

  THAssert(THCudaTensor_checkGPU(state, 6, input, output, weight,
                                 bias, columns, ones));
  luaL_argcheck(L, input->nDimension == 5, 2, "5D (batch mode) tensor is expected");
  luaL_argcheck(L, kH == kW && pH == pW, 2, "kH == kW && pH == pW is expected");

  // Batch size
  long batchSize = input->size[0];

  // Figure out the dimensions for individual gemms.
  int M_ = nInputPlane;
  int K_ = nOutputPlane * kT * kH * kW;
  int N_ = inputDepth * inputHeight * inputWidth;
  int N0_ = outputDepth * outputHeight * outputWidth;

  // Resize output
  THCudaTensor_resize5d(state, output, batchSize, nOutputPlane, outputDepth,
                        outputHeight, outputWidth);

  // Resize temporary columns
  THCudaTensor_resize5d(state, columns, 1, nOutputPlane * kT * kH * kW, inputDepth, inputHeight, inputWidth);

  // Define a buffer of ones, for bias accumulation
  // Note: this buffer can be shared with other modules, it only ever gets increased,
  // and always contains ones.
  if (ones->nDimension != 3 ||
    ones->size[0] * ones->size[1] * ones->size[2] < outputDepth * outputHeight * outputWidth) {
    // Resize plane and fill with ones...
    THCudaTensor_resize3d(state, ones, outputDepth, outputHeight, outputWidth);
    THCudaTensor_fill(state, ones, 1);
  }

  // Helpers
  THCudaTensor *input_n = THCudaTensor_new(state);
  THCudaTensor *output_n = THCudaTensor_new(state);

  for (int n = 0; n < batchSize; ++n) {
    THCudaTensor_select(state, input_n, input, 0, n);
    THCudaTensor_select(state, output_n, output, 0, n);

    // do gemm
    THCudaBlas_gemm(state, 'n', 't', N_, K_, M_,
    1, THCudaTensor_data(state, input_n), N_,
    THCudaTensor_data(state, weight), K_,
    0, THCudaTensor_data(state, columns), N_);

    // col2vol from columns -> output
    col2vol<float>(THCudaTensor_data(state, columns), nOutputPlane, outputDepth, outputHeight, outputWidth,
    kH, kT, pH, pT, dH, dT,
    THCudaTensor_data(state, output_n));

    // third, add bias
    THCudaBlas_gemm(state, 'n', 'n', N0_, nOutputPlane,
    1, 1,
    THCudaTensor_data(state, ones), N0_,
    THCudaTensor_data(state, bias), 1,
    1, THCudaTensor_data(state, output_n), N0_);

  }
  THCudaTensor_free(state, input_n);
  THCudaTensor_free(state, output_n);

  // return output
  return 1;
}

static int cunn_VolumetricFullConvolution_updateGradInput(lua_State *L) {
  THCState *state = getCutorchState(L);

  // Inputs
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");

  // Params
  int dT = luaT_getfieldcheckint(L, 1, "dT");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int kT = luaT_getfieldcheckint(L, 1, "kT");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int pT = luaT_getfieldcheckint(L, 1, "pT");
  int pH = luaT_getfieldcheckint(L, 1, "pH");
  int pW = luaT_getfieldcheckint(L, 1, "pW");
  int nInputPlane = luaT_getfieldcheckint(L, 1, "nInputPlane");
  int nOutputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");

  THCudaTensor *weight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *gradColumns = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "finput", "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");

  THAssert(THCudaTensor_checkGPU(state, 5, input, gradOutput, weight,
                                 gradColumns, gradInput));
  luaL_argcheck(L, input->nDimension == 5, 2, "5D (batch mode) tensor is expected");
  luaL_argcheck(L, kH == kW && pH == pW, 2, "kH == kW && pH == pW is expected");

  int inputDepth   = input->size[2];
  int inputHeight  = input->size[3];
  int inputWidth   = input->size[4];

  int outputDepth  = (inputDepth - 1) * dT - 2 * pT + kT;
  int outputHeight = (inputHeight - 1) * dH - 2 * pH + kH;
  int outputWidth  = (inputWidth - 1) * dW - 2 * pW + kW;

  // Batch size
  int batchSize = input->size[0];

  // Figure out the dimensions for individual gemms.
  int M_ = nInputPlane;
  int K_ = nOutputPlane * kT * kH * kW;
  int N_ = inputDepth * inputHeight * inputWidth;

  // Resize output
  THCudaTensor_resize5d(state, gradInput, batchSize, nInputPlane, inputDepth, inputHeight, inputWidth);

  // Resize temporary columns
  THCudaTensor_resize5d(state, gradColumns, 1, nOutputPlane * kT * kH * kW, inputDepth, inputHeight, inputWidth);

  // Helpers
  THCudaTensor *gradInput_n = THCudaTensor_new(state);
  THCudaTensor *gradOutput_n = THCudaTensor_new(state);

  // For each n in batch, do:
  for (int n = 0; n < batchSize; n++) {
    THCudaTensor_select(state, gradInput_n, gradInput, 0, n);
    THCudaTensor_select(state, gradOutput_n, gradOutput, 0, n);

    // vol2col from gradOutput to gradColumns
    vol2col<float>(THCudaTensor_data(state, gradOutput_n),
          nOutputPlane, outputDepth, outputHeight, outputWidth,
          kH, kT, pH, pT, dH, dT, THCudaTensor_data(state, gradColumns));

    // gemm to compute gradInput
    THCudaBlas_gemm(state, 'n', 'n', N_, M_, K_,
    				  1, THCudaTensor_data(state, gradColumns), N_,
              THCudaTensor_data(state, weight), K_,
    				  0, THCudaTensor_data(state, gradInput_n), N_);
  }

  // Free
  THCudaTensor_free(state, gradInput_n);
  THCudaTensor_free(state, gradOutput_n);

  // Return gradInput
  return 1;
}

static int cunn_VolumetricFullConvolution_accGradParameters(lua_State *L) {
  THCState *state = getCutorchState(L);

  // Inputs
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");

  // Params
  int dT = luaT_getfieldcheckint(L, 1, "dT");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int kT = luaT_getfieldcheckint(L, 1, "kT");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int pT = luaT_getfieldcheckint(L, 1, "pT");
  int pH = luaT_getfieldcheckint(L, 1, "pH");
  int pW = luaT_getfieldcheckint(L, 1, "pW");
  int nInputPlane = luaT_getfieldcheckint(L, 1, "nInputPlane");
  int nOutputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");

  THCudaTensor *gradWeight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradWeight", "torch.CudaTensor");
  THCudaTensor *gradBias = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradBias", "torch.CudaTensor");
  THCudaTensor *gradColumns = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "finput", "torch.CudaTensor");
  THCudaTensor *ones = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "fgradInput", "torch.CudaTensor");

  THAssert(THCudaTensor_checkGPU(state, 6, input, gradOutput, gradWeight,
                                 gradBias, gradColumns, ones));
  luaL_argcheck(L, input->nDimension == 5, 2, "5D (batch mode) tensor is expected");
  luaL_argcheck(L, kH == kW && pH == pW, 2, "kH == kW && pH == pW is expected");

  THCudaTensor_resize1d(state, gradBias, nOutputPlane);
  THCudaTensor_resize5d(state, gradWeight, nOutputPlane, nInputPlane, kT, kH, kW);

  int inputDepth   = input->size[2];
  int inputHeight  = input->size[3];
  int inputWidth   = input->size[4];

  int outputDepth  = (inputDepth - 1) * dT - 2 * pT + kT;
  int outputHeight = (inputHeight - 1) * dH - 2 * pH + kH;
  int outputWidth  = (inputWidth - 1) * dW - 2 * pW + kW;

  // Batch size
  long batchSize = input->size[0];

  // Figure out the dimensions for individual gemms.
  int M_ = nInputPlane;
  int K_ = nOutputPlane * kT * kH * kW;
  int N_ = inputDepth * inputHeight * inputWidth;
  int N0_ = outputDepth * outputHeight * outputWidth;

  // Resize temporary columns
  THCudaTensor_resize5d(state, gradColumns, 1, nOutputPlane * kT * kH * kW, inputDepth, inputHeight, inputWidth);

  if (ones->nDimension != 3 ||
    ones->size[0] * ones->size[1] * ones->size[2] < outputDepth * outputHeight * outputWidth) {
    // Resize plane and fill with ones...
    THCudaTensor_resize3d(state, ones, outputDepth, outputHeight, outputWidth);
    THCudaTensor_fill(state, ones, 1);
  }

  // Helpers
  THCudaTensor *input_n = THCudaTensor_new(state);
  THCudaTensor *gradOutput_n = THCudaTensor_new(state);

  // reset gradBias = 0
  CUDA_CHECK(L, hipMemset(THCudaTensor_data(state, gradBias), 0,
      sizeof(float) * nOutputPlane));
  // reset gradWeight = 0
  CUDA_CHECK(L, hipMemset(THCudaTensor_data(state, gradWeight), 0,
              sizeof(float) * M_ * K_));

  // For each n in batch, do:
  for (int n = 0; n < batchSize; n++) {
    THCudaTensor_select(state, input_n, input, 0, n);
    THCudaTensor_select(state, gradOutput_n, gradOutput, 0, n);

    // accumulate gradBias
    THCudaBlas_gemv(state, 't', N0_, nOutputPlane, 1,
                    THCudaTensor_data(state, gradOutput_n), N0_,
  	                THCudaTensor_data(state, ones), 1,
                    1,
  	                THCudaTensor_data(state, gradBias), 1);

    vol2col<float>(THCudaTensor_data(state, gradOutput_n),
          nOutputPlane, outputDepth, outputHeight, outputWidth,
          kH, kT, pH, pT, dH, dT, THCudaTensor_data(state, gradColumns));

    // accummulate gradWeight
    THCudaBlas_gemm(state, 't', 'n', K_, M_, N_,
          1, THCudaTensor_data(state, gradColumns), N_,
          THCudaTensor_data(state, input_n), N_,
          1, THCudaTensor_data(state, gradWeight), K_);
  }

  // Free
  THCudaTensor_free(state, input_n);
  THCudaTensor_free(state, gradOutput_n);

  // Return nothing
  return 0;
}

static const struct luaL_Reg cunn_VolumetricFullConvolution__ [] = {
  {"VolumetricFullConvolution_updateOutput", cunn_VolumetricFullConvolution_updateOutput},
  {"VolumetricFullConvolution_updateGradInput", cunn_VolumetricFullConvolution_updateGradInput},
  {"VolumetricFullConvolution_accGradParameters", cunn_VolumetricFullConvolution_accGradParameters},
  {NULL, NULL}
};

void cunn_VolumetricFullConvolution_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_VolumetricFullConvolution__, "nn");
  lua_pop(L,1);
}
